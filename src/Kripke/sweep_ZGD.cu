#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "Directions.h"


#define KRESTRICT __restrict__

//#define USE_PSI_HOST_MEM

//#define CU_TIMING

#define MAX ((a<b)?b:a)
#define MIN ((a>b)?b:a)


//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                              \
  hipError_t e=hipGetLastError();                                     \
  if(e!=hipSuccess) {                                                  \
    printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e)); \
    exit(EXIT_FAILURE);                                                 \
  }                                                                     \
}



__global__ void  LTimes_ZGD(double *phi, double * __restrict__ psi, const double * __restrict__ ell,
                                int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx);

__global__ void  LPlusTimes_ZGD(double *rhs, double * __restrict__ phi_out, double * __restrict__ ell_plus,
                                int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx);

__global__ void scattering_ZGD(int * __restrict__ d_mixed_to_zones, int * __restrict__ d_mixed_material,
                               double * __restrict__ d_mixed_fraction, int * __restrict__ d_mixed_offset,
                               double * __restrict__ d_phi, double *d_phi_out, double * __restrict__ d_sigs0,
                              double * __restrict__ d_sigs1, double * __restrict__ d_sigs2,
                               int * __restrict__ moment_to_coeff, int num_mixed, int num_moments, int num_groups, int num_coeff);

__global__ void scattering_ZGD_step2(const int * __restrict__ mixed_to_zones, const int * __restrict__ mixed_material,
                               const double * __restrict__ mixed_fraction, const int * __restrict__ mixed_offset,
                               const double * __restrict__ phi, double *phi_out, double * __restrict__ sigs0,
                               const double * __restrict__ sigs1, const double * __restrict__ sigs2,
                               const int * __restrict__ moment_to_coeff, int num_mixed, int num_moments, int num_groups, int num_coeff);


__global__ void scattering_ZGD_step3(const int * __restrict__ mixed_to_zones, const int * __restrict__ mixed_material,
                               const double * __restrict__ mixed_fraction, const int * __restrict__ mixed_offset,
                               const double * __restrict__ phi, double *phi_out, double * __restrict__ sigs0,
                               const double * __restrict__ sigs1, const double * __restrict__ sigs2,
                               const int * __restrict__ moment_to_coeff, int num_mixed, int num_moments, int num_groups, int num_coeff);


__global__ void source_ZGD(int * __restrict__ mixed_to_zones, int * __restrict__ mixed_material,
                               double * __restrict__ mixed_fraction, int * __restrict__ mixed_offset,
                               double *phi_out, int num_moments, int num_groups);

__global__ void  sweep_over_hyperplane_ZGD(int sliceID, int * __restrict__ offset, int * __restrict__ ii_jj_kk_z_idx, int num_directions, int num_groups,
                    int local_imax, int local_jmax, int local_kmax, 
                    double * __restrict__ dx, double * __restrict__ dy, double * __restrict__ dz, 
                    double * __restrict__ rhs, double * __restrict__ phi, double * __restrict__ psi, 
                    const double * __restrict__ sigt, const Directions * __restrict__ direction, 
                    double *i_plane, double *j_plane, double *k_plane);


__global__ void LPlusTimes_sweep_over_hyperplane_ZGD(int sliceID, int * __restrict__ offset, int * __restrict__ ii_jj_kk_z_idx, int num_directions, int num_groups, int nidx,
                    int local_imax, int local_jmax, int local_kmax, double * __restrict__ dx, double * __restrict__ dy,
                    double * __restrict__ dz, double *__restrict__ phi_out, double * __restrict__ ell_plus, double * __restrict__ psi,
                    const double * __restrict__ sigt, const Directions * __restrict__ direction,
                    double *i_plane, double *j_plane, double *k_plane);
					
					
					
int cuda_LTimes_ZGD(double *d_phi, double *h_psi, double *d_ell,
                    int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx);


int  cuda_LPlusTimes_ZGD(double *d_rhs, double *h_phi_out, double *h_ell_plus,
                    int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx);

int  cuda_scattering_ZGD(int *d_mixed_to_zones, int *d_mixed_material, double *d_mixed_fraction, int *d_mixed_offset,
                      double *d_phi, double *d_phi_out, double *d_sigs0,
                      double *d_sigs1, double *d_sigs2,
                      int *moment_to_coeff, int num_mixed, int num_moments, int num_groups, int num_coeff);

int  cuda_source_ZGD(int *d_mixed_to_zones, int *d_mixed_material, double *d_mixed_fraction, int *d_mixed_offset, 
                     double *d_phi_out, int num_moments,  int num_groups);


int cuda_sweep_ZGD( double *rhs, double *phi, double *psi,  double *sigt, Directions *direction,
                    double *i_plane, double *j_plane, double *k_plane,
                    int *ii_jj_kk_z_idx, int *offset, int *d_offset, double *dx, double *dy, double *dz,
                    int num_zones, int num_directions, int num_groups,
                    int local_imax, int local_jmax, int local_kmax, int Nslices);
int cuda_LPlusTimes_sweep_ZGD( double *phi_out, double *ell_plus,
                    double *psi, double *sigt,  Directions *direction,
                    double *i_plane, double *j_plane, double *k_plane,
                    int *ii_jj_kk_z_idx, int *h_offset, int *d_offset, double *dx, double *dy, double *dz,
                    int num_zones, int num_directions, int num_groups, int nidx,
                    int local_imax, int local_jmax, int local_kmax, int Nslices);


int cuda_LTimes_ZGD(double *d_phi, double *h_psi, double *d_ell,
                    int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx){

  cudaCheckError();
  int dim_y = 4;  
  dim3 threadsPerBlock(32,dim_y);

  LTimes_ZGD<<<num_zones,threadsPerBlock,nidx*dim_y*sizeof(double)>>>(d_phi,h_psi,d_ell,num_zones,num_groups,num_local_directions,num_local_groups,nidx);

  cudaCheckError();

  return 0;
}

__global__ void  LTimes_ZGD(double *phi, double * __restrict__ psi, const double * __restrict__ ell,
                                int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx){


      extern __shared__ double ss_phi[];

      int z = blockIdx.x;
      double * __restrict__ block_phi = &phi[z*num_groups*nidx];
      double * __restrict__ block_psi = &psi[z*num_local_groups*num_local_directions];

#if 1
     for(int group = threadIdx.y ;group < num_local_groups; group += blockDim.y){

        double *ss_phi_group = &ss_phi[nidx*threadIdx.y];

        for(int nm_offset = threadIdx.x; nm_offset < nidx; nm_offset+=blockDim.x)
           ss_phi_group[nm_offset] = block_phi[nm_offset+nidx*group];

        for (int d = 0; d < num_local_directions; d++) {

          double psi_d = block_psi[d+group*num_local_directions];

          for(int nm_offset = threadIdx.x; nm_offset < nidx; nm_offset+=blockDim.x)
            ss_phi_group[nm_offset] += ell[nm_offset + nidx*d] * psi_d;
        }

        for(int nm_offset = threadIdx.x; nm_offset < nidx; nm_offset+=blockDim.x)
           block_phi[nm_offset+nidx*group] =  ss_phi_group[nm_offset];
      }

#else

      for(int group = 0;group < num_local_groups;++ group){

        for(int nm_offset = threadIdx.x; nm_offset < nidx; nm_offset+=blockDim.x)
           ss_phi[nm_offset] = block_phi[nm_offset+nidx*group];
 
        for (int d = 0; d < num_local_directions; d++) {

          double psi_d = block_psi[d+group*num_local_directions];
           
          for(int nm_offset = threadIdx.x; nm_offset < nidx; nm_offset+=blockDim.x)
            ss_phi[nm_offset] += ell[nm_offset + nidx*d] * psi_d;          
        }

        for(int nm_offset = threadIdx.x; nm_offset < nidx; nm_offset+=blockDim.x)
           block_phi[nm_offset+nidx*group] =  ss_phi[nm_offset];

      }
#endif

}


/*******************/
int  cuda_scattering_ZGD(int *d_mixed_to_zones, int *d_mixed_material, double *d_mixed_fraction, int *d_mixed_offset,
                      double *d_phi, double *d_phi_out, double *d_sigs0, double *d_sigs1, double *d_sigs2,
                      int *d_moment_to_coeff, int num_mixed, int num_moments, int num_groups, int num_coeff){


     int y_dim = 6;
     dim3 threadsPerBlock(32,y_dim);


//     scattering_ZGD<<<480,threadsPerBlock>>>(d_mixed_to_zones,d_mixed_material,d_mixed_fraction,d_mixed_offset,
//                                d_phi,d_phi_out,d_sigs0, d_sigs1, d_sigs2, d_moment_to_coeff,num_mixed,num_moments,num_groups,num_coeff);


     scattering_ZGD_step3<<<480,threadsPerBlock,num_groups*y_dim*sizeof(double)>>>(d_mixed_to_zones,d_mixed_material,d_mixed_fraction,d_mixed_offset,
                                d_phi,d_phi_out,d_sigs0, d_sigs1, d_sigs2, d_moment_to_coeff,num_mixed,num_moments,num_groups,num_coeff);

     cudaCheckError();

    return 0;
}

/*******************/



__global__ void scattering_ZGD(int * __restrict__ mixed_to_zones, int * __restrict__ mixed_material,
                               double * __restrict__ mixed_fraction, int * __restrict__ mixed_offset,
                               double * __restrict__ phi, double *phi_out, double * __restrict__ sigs0,
                               double * __restrict__ sigs1, double * __restrict__ sigs2,
                               int * __restrict__ moment_to_coeff, int num_mixed, int num_moments, int num_groups, int num_coeff){


   int mix_min = mixed_offset[blockIdx.x];
   int mix_max = mixed_offset[blockIdx.x+1];
   double *d_sigs[3];
   d_sigs[0] = sigs0;
   d_sigs[1] = sigs1;
   d_sigs[2] = sigs2;

   for(int mix = mix_min;mix < mix_max; ++mix){
      int zone = mixed_to_zones[mix];
      int material = mixed_material[mix];
      double fraction = mixed_fraction[mix];
      double *sigs_g_gp = d_sigs[material];
      double *phi_z_g = &phi[zone*num_groups*num_moments];
      double *phi_out_z_gp = &phi_out[zone*num_groups*num_moments];

//LG unroll the outer loop
      for(int g = 0; g < num_groups;++g){

        for(int gp = threadIdx.y; gp < num_groups; gp += blockDim.y){

          for(int nm = threadIdx.x; nm < num_moments; nm += blockDim.x){
            // map nm to n
            int n = moment_to_coeff[nm];

            phi_out_z_gp[nm+gp*num_moments] += 
                sigs_g_gp[n + gp*num_coeff + g*num_groups*num_coeff] * 
                phi_z_g[nm + g*num_moments] * fraction;
          }
        }
      }

   }
}



__global__ void scattering_ZGD_step2(const int * __restrict__ mixed_to_zones, const int * __restrict__ mixed_material,
                               const double * __restrict__ mixed_fraction, const int * __restrict__ mixed_offset,
                               const double * __restrict__ phi, double *phi_out, double * __restrict__ sigs0,
                               const double * __restrict__ sigs1, const double * __restrict__ sigs2,
                               const int * __restrict__ moment_to_coeff, int num_mixed, int num_moments, int num_groups, int num_coeff){

   extern __shared__ double phi_z_g_ss[];

   int mix_min = mixed_offset[blockIdx.x];
   int mix_max = mixed_offset[blockIdx.x+1];
   int gid = threadIdx.x + threadIdx.y*blockDim.x;
   const double *d_sigs[3];
   d_sigs[0] = sigs0;
   d_sigs[1] = sigs1;
   d_sigs[2] = sigs2;

   for(int mix = mix_min;mix < mix_max; ++mix){
      int zone = mixed_to_zones[mix];
      int material = mixed_material[mix];
      double fraction = mixed_fraction[mix];
      const double * __restrict__ sigs_g_gp = d_sigs[material];
      const double * __restrict__ phi_z_g = &phi[zone*num_groups*num_moments];
      double * __restrict__ phi_out_z_gp = &phi_out[zone*num_groups*num_moments];

//LG unroll the outer loop
      for(int g = 0; g < num_groups;++g){

        __syncthreads();
        for(int nm = gid; nm < num_moments; nm += blockDim.x*blockDim.y)
          phi_z_g_ss[nm] =  phi_z_g[nm + g*num_moments]*fraction;
         __syncthreads(); 

        for(int gp = threadIdx.y; gp < num_groups; gp += blockDim.y){

          for(int nm = threadIdx.x; nm < num_moments; nm += blockDim.x){
            // map nm to n
            const int n = moment_to_coeff[nm];

            phi_out_z_gp[nm+gp*num_moments] +=
                sigs_g_gp[n + gp*num_coeff + g*num_groups*num_coeff] *
                phi_z_g_ss[nm];
          }
        }
      }

   }
}


__global__ void scattering_ZGD_step3(const int * __restrict__ mixed_to_zones, const int * __restrict__ mixed_material,
                               const double * __restrict__ mixed_fraction, const int * __restrict__ mixed_offset,
                               const double * __restrict__ phi, double *phi_out, double * __restrict__ sigs0,
                               const double * __restrict__ sigs1, const double * __restrict__ sigs2,
                               const int * __restrict__ moment_to_coeff, int num_mixed, int num_moments, int num_groups, int num_coeff){

   extern __shared__ double phi_out_ss[];

   int mix_min = mixed_offset[blockIdx.x];
   int mix_max = mixed_offset[blockIdx.x+1];
   const double *d_sigs[3];
   d_sigs[0] = sigs0;
   d_sigs[1] = sigs1;
   d_sigs[2] = sigs2;

   for(int mix = mix_min;mix < mix_max; ++mix){
      int zone = mixed_to_zones[mix];
      int material = mixed_material[mix];
      double fraction = mixed_fraction[mix];
      const double * __restrict__ sigs_g_gp = d_sigs[material];
      const double * __restrict__ phi_z_g = &phi[zone*num_groups*num_moments];
      double * __restrict__ phi_out_z_gp = &phi_out[zone*num_groups*num_moments];

//LG unroll the outer loop

      for(int gp = threadIdx.y; gp < num_groups; gp += blockDim.y){

        double *phi_out_ss_gp = &phi_out_ss[num_groups*threadIdx.y];
        for(int nm = threadIdx.x; nm < num_moments; nm += blockDim.x)
           phi_out_ss_gp[nm] =  phi_out_z_gp[nm+gp*num_moments];

        for(int g = 0; g < num_groups;++g){

          const int nm_shift = g*num_groups*num_coeff + gp*num_coeff ;
          for(int nm = threadIdx.x; nm < num_moments; nm += blockDim.x){
            // map nm to n
            const int n = moment_to_coeff[nm];

            phi_out_ss_gp[nm] +=
                sigs_g_gp[n + nm_shift] *
                phi_z_g[nm + g*num_moments]*fraction;

          }
        }
        for(int nm = threadIdx.x; nm < num_moments; nm += blockDim.x)
           phi_out_z_gp[nm+gp*num_moments] = phi_out_ss_gp[nm];

      }

   }
}







int  cuda_source_ZGD(int *d_mixed_to_zones, int *d_mixed_material, double *d_mixed_fraction, int *d_mixed_offset,
                      double *d_phi_out, int num_moments, int num_groups){

     dim3 threadsPerBlock(32,1);
     source_ZGD<<<480,threadsPerBlock>>>(d_mixed_to_zones,d_mixed_material,d_mixed_fraction,d_mixed_offset,
                                d_phi_out,num_moments,num_groups);

     cudaCheckError();
    return 0;
}

__global__ void source_ZGD(int * __restrict__ mixed_to_zones, int * __restrict__ mixed_material,
                               double * __restrict__ mixed_fraction, int * __restrict__ mixed_offset,
                               double *phi_out, int num_moments, int num_groups){


   int mix_min = mixed_offset[blockIdx.x];
   int mix_max = mixed_offset[blockIdx.x+1];


   for(int mix = mix_min;mix < mix_max; ++mix){
  
      int material = mixed_material[mix];

      if(material == 0){
          int zone = mixed_to_zones[mix];
          double fraction = mixed_fraction[mix];
          double *phi_out_z = &phi_out[zone*num_moments*num_groups];
          for(int g = threadIdx.x; g < num_groups; g += blockDim.x){
            phi_out_z[g*num_moments] += 1.0 * fraction;
          }
       }
    }

}


int  cuda_LPlusTimes_ZGD(double *d_rhs, double *d_phi_out, double *d_ell_plus,
                    int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx){


  #ifdef CU_TIMING__
  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float time_ms, time_s;

  hipEventRecord(start);
  #endif

  dim3 threadsPerBlock(32);

  LPlusTimes_ZGD<<<num_zones,threadsPerBlock,num_local_directions*sizeof(double)>>>(d_rhs,d_phi_out,d_ell_plus,num_zones,num_groups,num_local_directions,num_local_groups,nidx);

  #ifdef CU_TIMING__
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZGD: time to LPlusTimes_ZGD (GPU): %g [s]\n",time_s);
  #endif

  return 0;

}



__global__ void  LPlusTimes_ZGD(double *rhs, double * __restrict__ phi_out,
                                double * __restrict__ ell_plus,
                                int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx){


//LG consider running  for(int nm_offset  in parallel and using reduction within a warp

      int z = blockIdx.x;
      extern __shared__ double rhs_acc[];
      double *block_rhs =  &rhs[z*num_groups*num_local_directions];
      double *block_phi_out = &phi_out[z*num_groups*nidx];

      for(int group = 0; group < num_local_groups;++ group){

        for (int d = threadIdx.x; d < num_local_directions; d+=blockDim.x) {
          rhs_acc[d] = 0.0;

          for(int nm_offset = 0;nm_offset < nidx;++nm_offset)
             rhs_acc[d] += ell_plus[nm_offset + d*nidx] * block_phi_out[nm_offset + group*nidx];
          
          block_rhs[d+num_local_directions*group] += rhs_acc[d];
        }
      }
}


__global__ void sweep_over_hyperplane_ZGD_fluxRegisters ( const int nBlocks_j,
							  const int nBlocks_k,
							  const int i_inc,
							  const int j_inc,
							  const int k_inc,
							  const int direction_offset,
							  const int group, 
							  const int num_groups,
							  const int num_directions,
							  const int local_imax,
							  const int local_jmax,
							  const int local_kmax,
							  const double * __restrict__ d_dx, 
							  const double * __restrict__ d_dy, 
							  const double * __restrict__ d_dz, 
							  const double * __restrict__ d_rhs, 
							  const double * __restrict__ d_sigt, 
							  const Directions * __restrict__ d_direction,
							  double *d_psi, 
							  double * flux_boundary_i,
							  double * flux_boundary_j,
							  double * flux_boundary_k
							  )

/*

  Each block will process 16 directions per zone for 8 x 8 x local_imax zones 

  DRAM traffic is minimized by keeping x fluxes local to the thread (registers) and 
  sharing y and z fluxes using SMEM.

  smem_flux_j = 8 x 8 x 16 * 8 bytes = 8k Bytes arranged as dir / j / k 
  smem_flux_k = 8 x 8 x 16 * 8 bytes = 8k Bytes arranged as dir / j / k

*/

{

  int jBlock = 0;
  int kBlock = 0;

  // only handle octant 0 for now.  
  if ( i_inc != 0 || j_inc != 0 || k_inc != 0 ) return;

  // setup space to exchange j, k fluxes in SMEM  - per block
  extern __shared__ double smem[];
 
  //double * __volatile__ smem_flux_j = (double*) smem;                   // __volatile__ is required for cc10?
  //double * __volatile__  smem_flux_k = (double*) &smem_flux_j[1024];
  double * smem_flux_j = (double*) smem;
  double * smem_flux_k = (double*) &smem_flux_j[1024];

  int i = 0;                                                        // always start out at the i=0 plane

  const int tid = threadIdx.x ;                                     // local (i.e. rank) thread index

  const int d = tid%16 + direction_offset;                          // direction index (1/2 warp)

  // block index (i.e. within local_imax x 8 x 8 block)
  const int dd = tid%16;                                            // direction index (1/2 warp)
  const int jj = (tid/16)%8;                                        // j location within 8 x 8 plane
  const int kk = (tid/16)/8;                                        // k location within 8 x 8 plane

  // node index (i.e. within local_imax x local_jmax x local_kmax)
  int j = jBlock*8+jj;                                              // local (i.e. rank) y zone
  int k = kBlock*8+kk;                                              // local (i.e. rank) z zone

  double flux_i, flux_j, flux_k;

  // load this thread's constant data                               // i.e. constant for this zone-pencil in x
  const double xcos_dxi = d_direction [d].xcos * 2.0 / d_dx[i+1];   // zero effect on performance
  const double ycos_dyj = d_direction [d].ycos * 2.0 / d_dy[j+1];
  const double zcos_dzk = d_direction [d].zcos * 2.0 / d_dz[k+1];
  const int dir_grp = num_directions * num_groups;
  const int gd = d + group * num_directions;

  int z;       
  const double * block_sigt = NULL;  
  const double * block_rhs = NULL;    // pointer to rhs data
  double * __restrict__ block_psi;    // pointer to psi data

  // load in the incoming i-plane flux from GMEM for each thread
  flux_i = flux_boundary_i [dir_grp*(j+k*local_jmax) + group*num_directions + d ];

  // initialize the pointers to GMEM data
  z = j*local_imax + k*local_imax*local_jmax + i;
  block_sigt = &d_sigt[z*num_groups+group];
  block_rhs = &d_rhs[z*dir_grp + gd];    // pointer to rhs data
  block_psi = &d_psi[z*dir_grp + gd];    // pointer to psi data

  // loop over i-planes 
  // (a.k.a. loop over all the hyperplanes in the block allocated to the NODE)
  // This is done to minimize 'tail' effects. 8x8 pencils stack up with each other and leave zero gaps. 
  // Applies to all 8x8 pencils in the domain for this set of 16 directions.
  // A 32 x 32 node domain has 16 8x8 pencils.  Scaned linearly by j then by k.
  // Certainly, finding ways to parallelize this across multiple blocks would be a further optimization.
  for ( int hplane=0; hplane < local_imax*nBlocks_j*nBlocks_k + 16; ++hplane ) {

    // master sync - basically between hyperplanes - required to ensure that all flux data is in SMEM.
    // Significant performance limiter.  Removing this gives a 20% performance boost.
    __syncthreads();
    __threadfence_block();    // appears to help performance

    // check to see if the current zone is in the current hyperplane
    if ( kBlock < nBlocks_k && hplane >= jj + kk ) {
	
      if ( jj == 0 ) {
	// on the j-plane input boundary, so get flux_j from GMEM
	flux_j = flux_boundary_j [dir_grp*(i+k*local_imax) + group*num_directions + d]; 
      } 
      else {
	// get flux_j from one block to the side in SMEM
	flux_j = smem_flux_j[16*(kk+jj*8-8)+dd];
      }
      
      if ( kk == 0 ) {
	// on the k-plane input boundary, so get flux_k from GMEM
	flux_k = flux_boundary_k [dir_grp*(i+j*local_imax) + group*num_directions + d]; 
      }
      else {
	// get flux_k from the block in the lower row in SMEM
	flux_k = smem_flux_k [16*(jj+kk*8 - 8)+dd];
      }

      // calculate the new zonal flux
      double psi_z_g_d = ( ( 
			    //*block_rhs
			    __ldg(block_rhs)
			    + flux_i * xcos_dxi
			    + flux_j * ycos_dyj
			    + flux_k * zcos_dzk ) / 
			   ( 
			    //*block_sigt 
			    __ldg(block_sigt) 
			    + xcos_dxi 
			    + ycos_dyj  
			    + zcos_dzk )  
			   );

      // output new Psi to GMEM
      *block_psi = psi_z_g_d;

      psi_z_g_d *= 2;

      // update flux-i in register only
      flux_i = psi_z_g_d - flux_i;

      // update flux-j,k 
      flux_j = psi_z_g_d - flux_j;
      flux_k = psi_z_g_d - flux_k;

      // output flux boundaries
      if ( jj == 7 ) {
	// on the j-plane output boundary, so send flux_j to GMEM
	flux_boundary_j [dir_grp*(i+k*local_imax) + group*num_directions + d] = flux_j; 
      }
      else {
	// send flux_j to SMEM
	smem_flux_j[16*(kk+jj*8)+dd] = flux_j;
      }

      if ( kk == 7 ) {
	// on the k-plane output boundary, so send flux_k to GMEM
	flux_boundary_k [dir_grp*(i+j*local_imax) + group*num_directions + d] = flux_k; 
      }
      else {
	// send flux_k to SMEM
	smem_flux_k[16*(jj+kk*8)+dd] = flux_k;
      }

      // translate down the x-direction by 1
      i++;

      // update zone for input (rhs) and output (psi)
      block_rhs += dir_grp;
      block_psi += dir_grp;   
      block_sigt += num_groups;

      // handle reaching the end of the i-domain
      if ( i == local_imax ) {

	// at end of i-domain, send flux_i to GMEM
	flux_boundary_i [dir_grp*(j+k*local_jmax) + d + num_directions*group] = flux_i;  
 
	i = 0;                                           // i is reset
      	j += 8;                                          // increment to the same jj in the next j-block
	jBlock++;                                       
 
	// handle reaching the end of the j-domain
	if ( j >= local_jmax ) {
	  j = jj;
	  k += 8;
	  jBlock = 0;
	  kBlock++;
	}

	// update rhs, psi, sigt pointers to GMEM
	z = j*local_imax + k*local_imax*local_jmax + i;
	block_sigt = &d_sigt[z*num_groups+group];
	block_rhs = &d_rhs[z*dir_grp + gd];              // pointer to rhs data
	block_psi = &d_psi[z*dir_grp + gd];              // pointer to psi data

	// load the input flux_i for the new 8x8 block.
	flux_i = flux_boundary_i [dir_grp*(j+k*local_jmax) + group*num_directions + d ];
	
      }

    }

  }

}




int cuda_sweep_ZGD_fluxRegisters ( const int local_imax, 
				   const int local_jmax, 
				   const int local_kmax,
				   const int num_zones, 
				   const int num_directions, 
				   const int num_groups,
				   const double * __restrict__ d_rhs, 
				   const double * __restrict__ d_sigt,
				   const Directions * __restrict__ d_direction,
				   const double * __restrict__ d_dx,
				   const double * __restrict__ d_dy,
				   const double * __restrict__ d_dz,
				   double *h_psi, 
				   double *h_i_plane, 
				   double *h_j_plane, 
				   double *h_k_plane,
				   int i_inc,
				   int j_inc,
				   int k_inc
				   )

/* 

   Perform the sweep over the local zones while keeping the fluxes in register/smem.

   This removes the majority of the DRAM reads/writes (removes 6 of 8) and should
   give a corresponding performance improvement due to sweep being entirely bandwidth bound.

   By the time the code gets here, we know that all the boundary data for these local zones are known.

   This is a sweep over a single direction.  (ToDo: overlap with other directions.)

   This sweep occurs on a single GPU.  Expect this to be managed by a single rank.

   *** Requirements: *** 
      #directions must be a multiple of 16
      #zones in y and z must be a multiple of 8

   Each BLOCK handles 8 x 8 (j,k - zones) x 16 (directions) x 1 (energy group)

   One CUDA Block loops serially through j-k 'pencils' for 16 directions.
  
   Launch ngroups * ndirs / 16 number of KERNELS.
   
   Keep x-flux in register.

   Communicate y and z fluxes, within a CUDA block, via smem.  
      16k Bytes per block 

   Synchronize within a block using syncthreads();

   Synchronize between MPI ranks as current.

*/

{

  // Input Checks - since the current implementation has some restrictions on input parameters
  {
#ifdef USE_PSI_HOST_MEM
    printf ("calling cuda_sweep_ZGD_fluxRegisters with USE_PSI_HOST_MEM not supported (sensible).\n");
    abort();
#endif

#ifdef USE_IJK_PLANE_HOST_MEM
    printf ("calling cuda_sweep_ZGD_fluxRegisters with USE_IJK_PLANE_HOST_MEM not supported (sensible).\n");
    abort();
#endif

    if ( local_jmax%8 || local_kmax%8 ) {
      printf ("local y and z zone extents must be multiples of 8: %d, %d \n", local_jmax, local_kmax);
      abort();
    }
    
    if ( num_directions%16 ) {
      printf ("number of directions MUST be a multiple of 16 \n");
      abort();
    }
  }

  // Allocate space on the GPU for Psi and copy Psi to GPU.
  double *d_psi;
  size_t N = num_zones * num_directions * num_groups;
  {

#ifdef CU_TIMING
    hipEventRecord(start);
#endif

    hipMalloc((void **) &d_psi, N*sizeof(double));
    hipMemcpy(d_psi, h_psi,   N*sizeof(double), hipMemcpyHostToDevice);
    cudaCheckError();
    
#ifdef CU_TIMING
    hipEventRecord(stop);
    hipDeviceSynchronize();
    cudaCheckError();
    hipEventElapsedTime(&time_ms,start,stop);
    time_s=time_ms*.001;
    printf("ZGD: time to copy PSI H2D: %g [s]\n",time_s);
#endif
  }

  // Allocate space on the GPU for input fluxes.
  // *** NOTE that this had previously been arranged as ZDG.  However, it is advantageous
  //          to re-arrange as ZGD since we will always be reading 16 directions at once and
  //          this ZGD arrangements results in completely coalesced loads.  BUT this adds a
  //          lot of overhead to the CPU-portion which would need to be removed to get 
  //          legitimate performance numbers at scale.

  size_t groups_dirs = num_directions * num_groups;
  int i_plane_zones = local_jmax * local_kmax * groups_dirs;
  int j_plane_zones = local_imax * local_kmax * groups_dirs;
  int k_plane_zones = local_imax * local_jmax * groups_dirs; 

  double *d_i_plane,  *d_j_plane, *d_k_plane;

  #ifdef CU_TIMING
  hipEventRecord(start);
  #endif

  hipMalloc((void **) &d_i_plane, i_plane_zones * sizeof(double));
  hipMalloc((void **) &d_j_plane, j_plane_zones * sizeof(double));
  hipMalloc((void **) &d_k_plane, k_plane_zones * sizeof(double));

  double * tempi = (double *) malloc (i_plane_zones * sizeof(double));
  double * tempj = (double *) malloc (j_plane_zones * sizeof(double));
  double * tempk = (double *) malloc (k_plane_zones * sizeof(double));

  #pragma omp for
  for ( int k=0; k<local_kmax; k++ ) {
    for ( int j=0; j<local_jmax; j++ ) {
      for ( int g=0; g<num_groups; g++ ) {
	for ( int d=0; d<num_directions; d++ ) {
	  tempi[groups_dirs*(j+k*local_jmax)+g*num_directions+d] = h_i_plane[groups_dirs*(j+k*local_jmax)+d*num_groups+g];
	}
      }
    }
  }

  #pragma omp for
  for ( int k=0; k<local_kmax; k++ ) {
    for ( int i=0; i<local_imax; i++ ) {
      for ( int g=0; g<num_groups; g++ ) {
	for ( int d=0; d<num_directions; d++ ) {
	  tempj[groups_dirs*(i+k*local_imax)+g*num_directions+d] = h_j_plane[groups_dirs*(i+k*local_imax)+d*num_groups+g];
	}
      }
    }
  }

  #pragma omp for
  for ( int j=0; j<local_jmax; j++ ) {
    for ( int i=0; i<local_imax; i++ ) {
      for ( int g=0; g<num_groups; g++ ) {
	for ( int d=0; d<num_directions; d++ ) {
	  tempk[groups_dirs*(i+j*local_imax)+g*num_directions+d] = h_k_plane[groups_dirs*(i+j*local_imax)+d*num_groups+g];
	}
      }
    }
  }

  hipMemcpy(d_i_plane, tempi, i_plane_zones * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_j_plane, tempj, j_plane_zones * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_k_plane, tempk, k_plane_zones * sizeof(double), hipMemcpyHostToDevice);
  cudaCheckError();

  free(tempi);
  free(tempj);
  free(tempk);

  #ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZGD: time to copy ijk_PLANE H2D: %g [s]\n",time_s);
  #endif

  // number of required blocks
  int nBlocks_j = local_jmax/8;
  int nBlocks_k = local_kmax/8;

  // create streams
  hipStream_t sweepstreams[32];
  for ( int i=0; i<32; i++ ) {
    hipStreamCreate( &sweepstreams[i]);
  }
 
  // number of required kernels
  size_t nKernels = num_groups * (num_directions / 16);

  // just make sure that there are no dangling errors
  hipFuncSetCacheConfig(reinterpret_cast<const void*>( sweep_over_hyperplane_ZGD_fluxRegisters), hipFuncCachePreferShared );

  hipError_t cuerr2 = hipDeviceSetSharedMemConfig (hipSharedMemBankSizeEightByte);
  if (cuerr2) {
    abort();
  }

  int threadsPerBlock = 1024;                                                     // !fixed! 8 x 8 x 16

  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float time_ms, time_s;
  hipEventRecord(start);
  
  hipDeviceSynchronize();

  for ( int kernel=0; kernel<nKernels; kernel++ ) {

    //hipDeviceSynchronize();

    sweep_over_hyperplane_ZGD_fluxRegisters 
      <<< 1, threadsPerBlock, 16*1024, sweepstreams[kernel%32] >>> 
      ( nBlocks_j,
	nBlocks_k,
	i_inc,
	j_inc,
	k_inc,
	(kernel%(num_directions/16))*16,
	kernel/(num_directions/16),
	num_groups,
	num_directions,
	local_imax,
	local_jmax,
	local_kmax,
	d_dx, 
	d_dy, 
	d_dz, 
	d_rhs, 
	d_sigt, 
	d_direction,
	d_psi, 
	d_i_plane,                            
	d_j_plane,                            
	d_k_plane                             
	);
    
  }

  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;

  printf ("\nKernel Time = %e s \n", time_s);
  printf ("Bandwidth achieved = %e GB/s \n\n", 
	  1.0e-9 * (                                                                               // data from GMEM
		    8.0 * local_imax * local_jmax * local_kmax * num_directions * num_groups       // rhs - read
		    + 8.0 * local_imax * local_jmax * local_kmax * num_directions * num_groups     // psi - write
		    + 8.0 * local_jmax*local_kmax * num_directions * num_groups * 2                // flux_i - r/w
		    + 8.0 * local_imax*local_kmax * num_directions * num_groups * 2 * local_jmax/8 // flux_j - r/w
		    + 8.0 * local_imax*local_jmax * num_directions * num_groups * 2 * local_kmax/8 // flux_k - r/w
		    )
	  / time_s);
  
#ifdef CU_TIMING
  hipEventRecord(start);
#endif

  hipMemcpy(h_psi,     d_psi, N*sizeof(double),                   hipMemcpyDeviceToHost);

#ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZGD: time to copy PSI D2H: %g [s]\n",time_s);
#endif

  return 0;

}

int cuda_sweep_ZGD( double *d_rhs, double *h_phi, double *h_psi, double *d_sigt, Directions *d_direction,
                    double *h_i_plane, double *h_j_plane, double *h_k_plane,
                    int *d_ii_jj_kk_z_idx, int *h_offset,  int *d_offset, double *d_dx, double *d_dy, double *d_dz,
                    int num_zones, int num_directions, int num_groups,
                    int local_imax, int local_jmax, int local_kmax, int Nslices){

  size_t N;
  size_t groups_dirs;
  double *d_phi;
  float time_ms, time_s;
  

  N = num_zones * num_directions * num_groups;
  groups_dirs = num_directions * num_groups;


  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);



#ifdef USE_PSI_HOST_MEM
  double *d_psi = h_psi;
#else
  double *d_psi;
  #ifdef CU_TIMING
  hipEventRecord(start);
  #endif
  hipMalloc((void **) &d_psi, N*sizeof(double));
  hipMemcpy(d_psi, h_psi,   N*sizeof(double), hipMemcpyHostToDevice);
  cudaCheckError();

  #ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZGD: time to copy PSI H2D: %g [s]\n",time_s);
  #endif
#endif



  int i_plane_zones = local_jmax * local_kmax * groups_dirs;
  int j_plane_zones = local_imax * local_kmax * groups_dirs;
  int k_plane_zones = local_imax * local_jmax * groups_dirs;

#ifdef USE_IJK_PLANE_HOST_MEM

  double *d_i_plane = h_i_plane;
  double *d_j_plane = h_j_plane;
  double *d_k_plane = h_k_plane;

#else

  double *d_i_plane,  *d_j_plane, *d_k_plane;

  #ifdef CU_TIMING
  hipEventRecord(start);
  #endif

  hipMalloc((void **) &d_i_plane, i_plane_zones * sizeof(double));
  hipMalloc((void **) &d_j_plane, j_plane_zones * sizeof(double));
  hipMalloc((void **) &d_k_plane, k_plane_zones * sizeof(double));
  hipMemcpy(d_i_plane, h_i_plane, i_plane_zones * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_j_plane, h_j_plane, j_plane_zones * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_k_plane, h_k_plane, k_plane_zones * sizeof(double), hipMemcpyHostToDevice);
  cudaCheckError();

  #ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZGD: time to copy ijk_PLANE H2D: %g [s]\n",time_s);
  #endif

#endif

  cudaCheckError();
 

//  hipFuncSetCacheConfig(reinterpret_cast<const void*>(sweep_over_hyperplane_ZGD), hipFuncCachePreferL1);

//call cuda kernel to sweep over hyperplanes(slices)

  dim3 threadsPerBlock(32,8);

  for (int slice = 0; slice < Nslices; slice++){
    
     #ifdef CU_TIMING
     hipEventRecord(start);                                             
     #endif

     dim3 numBlocks = h_offset[slice+1] - h_offset[slice];
     sweep_over_hyperplane_ZGD<<<numBlocks,threadsPerBlock>>>(slice,d_offset,d_ii_jj_kk_z_idx,num_directions,num_groups,local_imax,local_jmax,local_kmax,
                                                          d_dx, d_dy, d_dz, d_rhs, d_phi, d_psi, d_sigt, d_direction, d_i_plane, d_j_plane, d_k_plane);
     #ifdef CU_TIMING
     hipEventRecord(stop);                                              
     hipDeviceSynchronize();                                            
     cudaCheckError();                                                   
     float time_ms, time_s;                                              
     hipEventElapsedTime(&time_ms,start,stop);                          
     time_s=time_ms*.001;                                                
     printf("ZGD: #blocks=%d, time=%g [s]\n",h_offset[slice+1] - h_offset[slice],time_s);
     #endif
     
  }

  #ifdef CU_TIMING
  hipEventRecord(start);
  #endif

#ifndef USE_IJK_PLANE_HOST_MEM
    hipMemcpy(h_i_plane, d_i_plane, i_plane_zones * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_j_plane, d_j_plane, j_plane_zones * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_k_plane, d_k_plane, k_plane_zones * sizeof(double), hipMemcpyDeviceToHost);
#endif

  #ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZGD: time to copy ijk_PLANE D2H: %g [s]\n",time_s);
  #endif


#ifndef USE_PSI_HOST_MEM

#ifdef CU_TIMING
  hipEventRecord(start);
#endif

  hipMemcpy(h_psi,     d_psi, N*sizeof(double),                   hipMemcpyDeviceToHost);

#ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZGD: time to copy PSI D2H: %g [s]\n",time_s);
#endif

  hipFree(d_psi);
#endif


//  hipFree(d_phi);

#ifndef USE_IJK_PLANE_HOST_MEM
  hipFree(d_i_plane);
  hipFree(d_j_plane);
  hipFree(d_k_plane);
#endif

  cudaCheckError();

  return 0;
}

#define I_PLANE_INDEX(j, k) ((k)*(local_jmax) + (j))
#define J_PLANE_INDEX(i, k) ((k)*(local_imax) + (i))
#define K_PLANE_INDEX(i, j) ((j)*(local_imax) + (i))


__global__ void  sweep_over_hyperplane_ZGD(int sliceID, int * __restrict__ offset, int * __restrict__ ii_jj_kk_z_idx, int num_directions, int num_groups,
                    int local_imax, int local_jmax, int local_kmax,
                    double * __restrict__ dx, double * __restrict__ dy, double * __restrict__ dz,
                    double * __restrict__ rhs, double * __restrict__ phi, double * __restrict__ psi,
                    const double * __restrict__ sigt, const Directions * __restrict__ direction,
                    double *i_plane, double *j_plane, double *k_plane){

 
//each block will process one element 
      int element = offset[sliceID] + blockIdx.x;
      if (element > offset[sliceID+1]) return; 

      int i    = ii_jj_kk_z_idx[element*4];
      int j    = ii_jj_kk_z_idx[element*4+1];
      int k    = ii_jj_kk_z_idx[element*4+2];
      int z    = ii_jj_kk_z_idx[element*4+3];
      int dir_grp = num_directions*num_groups;
      int I_P_I = I_PLANE_INDEX(j, k);
      int J_P_I = J_PLANE_INDEX(i, k);
      int K_P_I = K_PLANE_INDEX(i, j);
      double two_inv_dxi = 2.0/dx[i + 1];
      double two_inv_dyj = 2.0/dy[j + 1];
      double two_inv_dzk = 2.0/dz[k + 1];
     

      double * KRESTRICT  block_rhs = &rhs[z*dir_grp];
//    double * KRESTRICT  block_phi = &phi[z*num_directions*num_groups];
      double * KRESTRICT  block_psi = &psi[z*dir_grp];
      const double * KRESTRICT  block_sigt = &sigt[z*num_groups];

      double * KRESTRICT psi_lf_z = &i_plane[I_P_I*dir_grp]; 
      double * KRESTRICT psi_fr_z = &j_plane[J_P_I*dir_grp]; 
      double * KRESTRICT psi_bo_z = &k_plane[K_P_I*dir_grp]; 

     

      for (int group = threadIdx.y; group < num_groups; group += blockDim.y){

          for (int  d = threadIdx.x; d < num_directions; d += blockDim.x){

            int gd = d + group*num_directions;

            double xcos_dxi =  direction[d].xcos * two_inv_dxi; 
            double ycos_dyj =  direction[d].ycos * two_inv_dyj;
            double zcos_dzk =  direction[d].zcos * two_inv_dzk;

            double psi_lf_z_g_d = psi_lf_z[gd];
            double psi_fr_z_g_d = psi_fr_z[gd];
            double psi_bo_z_g_d = psi_bo_z[gd];

            /* Calculate new zonal flux */
            double psi_z_g_d = (block_rhs[gd]
                + psi_lf_z_g_d * xcos_dxi
                + psi_fr_z_g_d * ycos_dyj
                + psi_bo_z_g_d * zcos_dzk)
                / (xcos_dxi + ycos_dyj + zcos_dzk + block_sigt[group]);

            block_psi[gd] = psi_z_g_d;


            /* Apply diamond-difference relationships */
            psi_lf_z[gd] = 2.0 * psi_z_g_d - psi_lf_z_g_d;
            psi_fr_z[gd] = 2.0 * psi_z_g_d - psi_fr_z_g_d;
            psi_bo_z[gd] = 2.0 * psi_z_g_d - psi_bo_z_g_d;
          }
        }
}

int cuda_LPlusTimes_sweep_ZGD( double *d_phi_out, double *d_ell_plus, double *h_psi, double *d_sigt, Directions *d_direction,
                    double *h_i_plane, double *h_j_plane, double *h_k_plane,
                    int *d_ii_jj_kk_z_idx, int *h_offset,  int *d_offset, double *d_dx, double *d_dy, double *d_dz,
                    int num_zones, int num_directions, int num_groups, int nidx,
                    int local_imax, int local_jmax, int local_kmax, int Nslices){


  size_t N;
  size_t groups_dirs;
  float time_ms, time_s;
  
  N = num_zones * num_directions * num_groups;
  groups_dirs = num_directions * num_groups;

  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);


#ifdef USE_PSI_HOST_MEM
  double *d_psi = h_psi;
#else
  double *d_psi;
  #ifdef CU_TIMING
  hipEventRecord(start);
  #endif
  hipMalloc((void **) &d_psi, N*sizeof(double));
  hipMemcpy(d_psi, h_psi,   N*sizeof(double), hipMemcpyHostToDevice);
  cudaCheckError();

  #ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZGD: time to copy PSI H2D: %g [s]\n",time_s);
  #endif
#endif

  int i_plane_zones = local_jmax * local_kmax * groups_dirs;
  int j_plane_zones = local_imax * local_kmax * groups_dirs;
  int k_plane_zones = local_imax * local_jmax * groups_dirs;

#ifdef USE_IJK_PLANE_HOST_MEM

  double *d_i_plane = h_i_plane;
  double *d_j_plane = h_j_plane;
  double *d_k_plane = h_k_plane;

#else

  double *d_i_plane,  *d_j_plane, *d_k_plane;

  #ifdef CU_TIMING
  hipEventRecord(start);
  #endif

  hipMalloc((void **) &d_i_plane, i_plane_zones * sizeof(double));
  hipMalloc((void **) &d_j_plane, j_plane_zones * sizeof(double));
  hipMalloc((void **) &d_k_plane, k_plane_zones * sizeof(double));
  hipMemcpy(d_i_plane, h_i_plane, i_plane_zones * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_j_plane, h_j_plane, j_plane_zones * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_k_plane, h_k_plane, k_plane_zones * sizeof(double), hipMemcpyHostToDevice);
  cudaCheckError();

  #ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZGD: time to copy ijk_PLANE H2D: %g [s]\n",time_s);
  #endif

#endif

  cudaCheckError();
 

  hipFuncSetCacheConfig(reinterpret_cast<const void*>(sweep_over_hyperplane_ZGD), hipFuncCachePreferL1);

//call cuda kernel to sweep over hyperplanes(slices)
  int dim_y = 8; 
  dim3 threadsPerBlock(32,dim_y);

  for (int slice = 0; slice < Nslices; slice++){
    
     #ifdef CU_TIMING
     hipEventRecord(start);                                             
     #endif

     dim3 numBlocks = h_offset[slice+1] - h_offset[slice];
     LPlusTimes_sweep_over_hyperplane_ZGD<<<numBlocks,threadsPerBlock,num_directions*dim_y*sizeof(double)>>>(slice,d_offset,d_ii_jj_kk_z_idx,num_directions,num_groups,nidx,local_imax,local_jmax,local_kmax,
                                                          d_dx, d_dy, d_dz, d_phi_out, d_ell_plus, d_psi, d_sigt, d_direction, d_i_plane, d_j_plane, d_k_plane);
     #ifdef CU_TIMING
     hipEventRecord(stop);                                              
     hipDeviceSynchronize();                                            
     cudaCheckError();                                                   
     float time_ms, time_s;                                              
     hipEventElapsedTime(&time_ms,start,stop);                          
     time_s=time_ms*.001;                                                
     printf("ZGD: #blocks=%d, time=%g [s]\n",h_offset[slice+1] - h_offset[slice],time_s);
     #endif
     
  }

  #ifdef CU_TIMING
  hipEventRecord(start);
  #endif

#ifndef USE_IJK_PLANE_HOST_MEM
    hipMemcpy(h_i_plane, d_i_plane, i_plane_zones * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_j_plane, d_j_plane, j_plane_zones * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_k_plane, d_k_plane, k_plane_zones * sizeof(double), hipMemcpyDeviceToHost);
#endif

  #ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZGD: time to copy ijk_PLANE D2H: %g [s]\n",time_s);
  #endif


#ifndef USE_PSI_HOST_MEM

#ifdef CU_TIMING
  hipEventRecord(start);
#endif

  hipMemcpy(h_psi,     d_psi, N*sizeof(double),                   hipMemcpyDeviceToHost);

#ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZGD: time to copy PSI D2H: %g [s]\n",time_s);
#endif

  hipFree(d_psi);
#endif


//  hipFree(d_phi);

#ifndef USE_IJK_PLANE_HOST_MEM
  hipFree(d_i_plane);
  hipFree(d_j_plane);
  hipFree(d_k_plane);
#endif

  cudaCheckError();


  return 0;
}


__global__ void LPlusTimes_sweep_over_hyperplane_ZGD(int sliceID, int * __restrict__ offset, int * __restrict__ ii_jj_kk_z_idx, int num_directions, int num_groups, int nidx,
                    int local_imax, int local_jmax, int local_kmax, double * __restrict__ dx, double * __restrict__ dy, 
                    double * __restrict__ dz, double *__restrict__ phi_out, double * __restrict__ ell_plus, double * __restrict__ psi, 
                    const double * __restrict__ sigt, const Directions * __restrict__ direction,
                    double *i_plane, double *j_plane, double *k_plane){
 
     extern __shared__ double rhs_local[];
 
//each block will process one element 
      int element = offset[sliceID] + blockIdx.x;
      if (element > offset[sliceID+1]) return; 

      int i    = ii_jj_kk_z_idx[element*4];
      int j    = ii_jj_kk_z_idx[element*4+1];
      int k    = ii_jj_kk_z_idx[element*4+2];
      int z    = ii_jj_kk_z_idx[element*4+3];
      int dir_grp = num_directions*num_groups;
      int I_P_I = I_PLANE_INDEX(j, k);
      int J_P_I = J_PLANE_INDEX(i, k);
      int K_P_I = K_PLANE_INDEX(i, j);
      double two_inv_dxi = 2.0/dx[i + 1];
      double two_inv_dyj = 2.0/dy[j + 1];
      double two_inv_dzk = 2.0/dz[k + 1];
     
      double * KRESTRICT  block_psi = &psi[z*dir_grp];
      const double * KRESTRICT  block_sigt = &sigt[z*num_groups];
      double * KRESTRICT psi_lf_z = &i_plane[I_P_I*dir_grp]; 
      double * KRESTRICT psi_fr_z = &j_plane[J_P_I*dir_grp]; 
      double * KRESTRICT psi_bo_z = &k_plane[K_P_I*dir_grp]; 
      const double * KRESTRICT block_phi_out = &phi_out[z*num_groups*nidx];

      for (int group = threadIdx.y; group < num_groups; group += blockDim.y){

	  double * __restrict__ rhs_local_group = &rhs_local[threadIdx.y * num_directions];
          for (int d = threadIdx.x; d < num_directions; d+=blockDim.x) {
            double sum  = 0.0;
            for(int nm_offset = 0;nm_offset < nidx;++nm_offset)
               sum += ell_plus[nm_offset + d*nidx] * block_phi_out[nm_offset + group*nidx];
            rhs_local_group[d] = sum;     
          }	  
	 
          for (int  d = threadIdx.x; d < num_directions; d += blockDim.x){

            int gd = d + group*num_directions;

            double xcos_dxi =  direction[d].xcos * two_inv_dxi; 
            double ycos_dyj =  direction[d].ycos * two_inv_dyj;
            double zcos_dzk =  direction[d].zcos * two_inv_dzk;

            double psi_lf_z_g_d = psi_lf_z[gd];
            double psi_fr_z_g_d = psi_fr_z[gd];
            double psi_bo_z_g_d = psi_bo_z[gd];

            /* Calculate new zonal flux */
            double psi_z_g_d = (rhs_local_group[d]
                + psi_lf_z_g_d * xcos_dxi
                + psi_fr_z_g_d * ycos_dyj
                + psi_bo_z_g_d * zcos_dzk)
                / (xcos_dxi + ycos_dyj + zcos_dzk + block_sigt[group]);

            block_psi[gd] = psi_z_g_d;


            /* Apply diamond-difference relationships */
            psi_lf_z[gd] = 2.0 * psi_z_g_d - psi_lf_z_g_d;
            psi_fr_z[gd] = 2.0 * psi_z_g_d - psi_fr_z_g_d;
            psi_bo_z[gd] = 2.0 * psi_z_g_d - psi_bo_z_g_d;
          }
        }
}
