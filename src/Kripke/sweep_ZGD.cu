#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "Directions.h"


#define KRESTRICT __restrict__

#define USE_PSI_HOST_MEM

//#define CU_TIMING

#define MAX ((a<b)?b:a)
#define MIN ((a>b)?b:a)


//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                              \
  hipError_t e=hipGetLastError();                                     \
  if(e!=hipSuccess) {                                                  \
    printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e)); \
    exit(EXIT_FAILURE);                                                 \
  }                                                                     \
}


__global__ void  LTimes_ZGD(double *phi, double * __restrict__ psi, double * __restrict__ ell,
                                int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx);


__global__ void  LPlusTimes_ZGD(double *rhs, double * __restrict__ phi_out, double * __restrict__ ell_plus,
                                int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx);



__global__ void  sweep_over_hyperplane_ZGD(int sliceID, int *offset, int *ii_jj_kk_z_idx, int num_directions, int num_groups,
                    int local_imax, int local_jmax, int local_kmax, double *dx, double *dy, double *dz, double *rhs, double *phi, double *psi, 
                    double *sigt, Directions *direction, 
                    double *i_plane, double *j_plane, double *k_plane);


int cuda_LTimes_ZGD(double *d_phi, double *h_psi, double *d_ell,
                    int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx);


int  cuda_LPlusTimes_ZGD(double *d_rhs, double *h_phi_out, double *h_ell_plus,
                    int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx);

int cuda_sweep_ZGD( double *rhs, double *phi, double *psi,  double *sigt, Directions *direction,
                    double *i_plane, double *j_plane, double *k_plane,
                    int *ii_jj_kk_z_idx, int *offset, int *d_offset, double *dx, double *dy, double *dz,
                    int num_zones, int num_directions, int num_groups,
                    int local_imax, int local_jmax, int local_kmax, int Nslices);


int cuda_LTimes_ZGD(double *d_phi, double *h_psi, double *d_ell,
                    int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx){

  cudaCheckError();

  dim3 threadsPerBlock(32);

  LTimes_ZGD<<<num_zones,threadsPerBlock,nidx*sizeof(double)>>>(d_phi,h_psi,d_ell,num_zones,num_groups,num_local_directions,num_local_groups,nidx);

  cudaCheckError();

  return 0;
}

__global__ void  LTimes_ZGD(double *phi, double * __restrict__ psi, double * __restrict__ ell,
                                int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx){


      extern __shared__ double ss_phi[];

      int z = blockIdx.x;
      double *block_phi = &phi[z*num_groups*nidx];
      double *block_psi = &psi[z*num_local_groups*num_local_directions];


      for(int group = 0;group < num_local_groups;++ group){
         

        for(int nm_offset = threadIdx.x; nm_offset < nidx; nm_offset+=blockDim.x)
           ss_phi[nm_offset] = block_phi[nm_offset+nidx*group];
 
        for (int d = 0; d < num_local_directions; d++) {

          double psi_d = block_psi[d+group*num_local_directions];
           
          for(int nm_offset = threadIdx.x; nm_offset < nidx; nm_offset+=blockDim.x)
            ss_phi[nm_offset] += ell[nm_offset + nidx*d] * psi_d;          
        }
        for(int nm_offset = threadIdx.x; nm_offset < nidx; nm_offset+=blockDim.x)
           block_phi[nm_offset+nidx*group] =  ss_phi[nm_offset];

      }

}



int  cuda_LPlusTimes_ZGD(double *d_rhs, double *h_phi_out, double *d_ell_plus,
                    int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx){


  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float time_ms, time_s;
  double *d_phi_out;

  #ifdef CU_TIMING
  hipEventRecord(start);
  #endif

  hipMalloc((void **) &d_phi_out,num_zones*nidx * num_groups * sizeof(double));
//  hipMalloc((void **) &d_ell_plus, nidx * num_local_directions * sizeof(double));
  hipMemcpy(d_phi_out, h_phi_out,num_zones * nidx * num_groups * sizeof(double), hipMemcpyHostToDevice);
//  cudaCheckError();
//  hipMemcpy(d_ell_plus, h_ell_plus, nidx * num_local_directions * sizeof(double), hipMemcpyHostToDevice);
  cudaCheckError();

  #ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZGD: time to copy d_phi_out+d_ell_plus H2D: %g [s]\n",time_s);
  #endif
  #ifdef CU_TIMING
  hipEventRecord(start);
  #endif

  dim3 threadsPerBlock(32);

  LPlusTimes_ZGD<<<num_zones,threadsPerBlock,num_local_directions*sizeof(double)>>>(d_rhs,d_phi_out,d_ell_plus,num_zones,num_groups,num_local_directions,num_local_groups,nidx);

  #ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZGD: time to LPlusTimes_ZGD (GPU): %g [s]\n",time_s);
  #endif


  hipFree(d_phi_out);
//  hipFree(d_ell_plus);

  return 0;

}



__global__ void  LPlusTimes_ZGD(double *rhs, double * __restrict__ phi_out,
                                double * __restrict__ ell_plus,
                                int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx){


      int z = blockIdx.x;
      extern __shared__ double rhs_acc[];
      double *block_rhs =  &rhs[z*num_groups*num_local_directions];
      double *block_phi_out = &phi_out[z*num_groups*nidx];

      for(int group = 0; group < num_local_groups;++ group){
        for (int d = threadIdx.x; d < num_local_directions; d+=blockDim.x) {
          rhs_acc[d] = 0.0;

          for(int nm_offset = 0;nm_offset < nidx;++nm_offset)
             rhs_acc[d] += ell_plus[nm_offset + d*nidx] * block_phi_out[nm_offset + group*nidx];
          
          block_rhs[d+num_local_directions*group] += rhs_acc[d];
        }
      }
}




int cuda_sweep_ZGD( double *d_rhs, double *h_phi, double *h_psi, double *d_sigt, Directions *d_direction,
                    double *h_i_plane, double *h_j_plane, double *h_k_plane,
                    int *d_ii_jj_kk_z_idx, int *h_offset,  int *d_offset, double *d_dx, double *d_dy, double *d_dz,
                    int num_zones, int num_directions, int num_groups,
                    int local_imax, int local_jmax, int local_kmax, int Nslices){


  size_t N;
  size_t groups_dirs;
  double *d_phi;
  float time_ms, time_s;
  

  N = num_zones * num_directions * num_groups;
  groups_dirs = num_directions * num_groups;


  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);



#ifdef USE_PSI_HOST_MEM
  double *d_psi = h_psi;
#else
  double *d_psi;
  #ifdef CU_TIMING
  hipEventRecord(start);
  #endif
  hipMalloc((void **) &d_psi, N*sizeof(double));
  hipMemcpy(d_psi, h_psi,   N*sizeof(double), hipMemcpyHostToDevice);
  cudaCheckError();

  #ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZGD: time to copy PSI H2D: %g [s]\n",time_s);
  #endif
#endif



  int i_plane_zones = local_jmax * local_kmax * groups_dirs;
  int j_plane_zones = local_imax * local_kmax * groups_dirs;
  int k_plane_zones = local_imax * local_jmax * groups_dirs;

#ifdef USE_IJK_PLANE_HOST_MEM

  double *d_i_plane = h_i_plane;
  double *d_j_plane = h_j_plane;
  double *d_k_plane = h_k_plane;

#else

  double *d_i_plane,  *d_j_plane, *d_k_plane;

  #ifdef CU_TIMING
  hipEventRecord(start);
  #endif

  hipMalloc((void **) &d_i_plane, i_plane_zones * sizeof(double));
  hipMalloc((void **) &d_j_plane, j_plane_zones * sizeof(double));
  hipMalloc((void **) &d_k_plane, k_plane_zones * sizeof(double));
  hipMemcpy(d_i_plane, h_i_plane, i_plane_zones * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_j_plane, h_j_plane, j_plane_zones * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_k_plane, h_k_plane, k_plane_zones * sizeof(double), hipMemcpyHostToDevice);
  cudaCheckError();

  #ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZGD: time to copy ijk_PLANE H2D: %g [s]\n",time_s);
  #endif

#endif




//  hipMalloc((void **) &d_phi, N*sizeof(double));

//  hipMemcpy(d_phi, h_phi,   N*sizeof(double), hipMemcpyHostToDevice);


  cudaCheckError();
 

  hipFuncSetCacheConfig(reinterpret_cast<const void*>(sweep_over_hyperplane_ZGD), hipFuncCachePreferL1);

//call cuda kernel to sweep over hyperplanes(slices)

  dim3 threadsPerBlock(32,4);

  for (int slice = 0; slice < Nslices; slice++){
    
     #ifdef CU_TIMING
     hipEventRecord(start);                                             
     #endif

     dim3 numBlocks = h_offset[slice+1] - h_offset[slice];
     sweep_over_hyperplane_ZGD<<<numBlocks,threadsPerBlock, num_groups*sizeof(double) >>>(slice,d_offset,d_ii_jj_kk_z_idx,num_directions,num_groups,local_imax,local_jmax,local_kmax,
                                                          d_dx, d_dy, d_dz, d_rhs, d_phi, d_psi, d_sigt, d_direction, d_i_plane, d_j_plane, d_k_plane);
     #ifdef CU_TIMING
     hipEventRecord(stop);                                              
     hipDeviceSynchronize();                                            
     cudaCheckError();                                                   
     float time_ms, time_s;                                              
     hipEventElapsedTime(&time_ms,start,stop);                          
     time_s=time_ms*.001;                                                
     printf("ZGD: #blocks=%d, time=%g [s]\n",h_offset[slice+1] - h_offset[slice],time_s);
     #endif
     
  }

  #ifdef CU_TIMING
  hipEventRecord(start);
  #endif

#ifndef USE_IJK_PLANE_HOST_MEM
    hipMemcpy(h_i_plane, d_i_plane, i_plane_zones * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_j_plane, d_j_plane, j_plane_zones * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_k_plane, d_k_plane, k_plane_zones * sizeof(double), hipMemcpyDeviceToHost);
#endif

  #ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZGD: time to copy ijk_PLANE D2H: %g [s]\n",time_s);
  #endif


#ifndef USE_PSI_HOST_MEM

#ifdef CU_TIMING
  hipEventRecord(start);
#endif

  hipMemcpy(h_psi,     d_psi, N*sizeof(double),                   hipMemcpyDeviceToHost);

#ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZGD: time to copy PSI D2H: %g [s]\n",time_s);
#endif

  hipFree(d_psi);
#endif


//  hipFree(d_phi);

#ifndef USE_IJK_PLANE_HOST_MEM
  hipFree(d_i_plane);
  hipFree(d_j_plane);
  hipFree(d_k_plane);
#endif

  cudaCheckError();

  return 0;
}

#define I_PLANE_INDEX(j, k) ((k)*(local_jmax) + (j))
#define J_PLANE_INDEX(i, k) ((k)*(local_imax) + (i))
#define K_PLANE_INDEX(i, j) ((j)*(local_imax) + (i))


__global__ void sweep_over_hyperplane_ZGD(int sliceID, int *offset, int *ii_jj_kk_z_idx, int num_directions, int num_groups,
                    int local_imax, int local_jmax, int local_kmax, double *dx, double *dy, double *dz, double *rhs, double *phi, double *psi, 
                    double *sigt, Directions *direction,
                    double *i_plane, double *j_plane, double *k_plane){


 
//each block will process one element 
      int element = offset[sliceID] + blockIdx.x;
      if (element > offset[sliceID+1]) return; 

      int i    = ii_jj_kk_z_idx[element*4];
      int j    = ii_jj_kk_z_idx[element*4+1];
      int k    = ii_jj_kk_z_idx[element*4+2];
      int z    = ii_jj_kk_z_idx[element*4+3];
      int dir_grp = num_directions*num_groups;
      int I_P_I = I_PLANE_INDEX(j, k);
      int J_P_I = J_PLANE_INDEX(i, k);
      int K_P_I = K_PLANE_INDEX(i, j);
      double two_inv_dxi = 2.0/dx[i + 1];
      double two_inv_dyj = 2.0/dy[j + 1];
      double two_inv_dzk = 2.0/dz[k + 1];
     

      double * KRESTRICT  block_rhs = &rhs[z*dir_grp];
//    double * KRESTRICT  block_phi = &phi[z*num_directions*num_groups];
      double * KRESTRICT  block_psi = &psi[z*dir_grp];
      double * KRESTRICT  block_sigt = &sigt[z*num_groups];

      double * KRESTRICT psi_lf_z = &i_plane[I_P_I*dir_grp]; 
      double * KRESTRICT psi_fr_z = &j_plane[J_P_I*dir_grp]; 
      double * KRESTRICT psi_bo_z = &k_plane[K_P_I*dir_grp]; 

//      extern __shared__ double xyzcos_dxyz[];

      
//      for( int d = threadIdx.x + threadIdx.y*blockDim.x;  d < num_directions; d = d + blockDim.x*blockDim.y){//not working - BUG?
//      for( int d = threadIdx.x;  d < num_directions; d = d + blockDim.x){
//         xyzcos_dxyz[3*d]   = direction[d].xcos * two_inv_dxi;
//         xyzcos_dxyz[3*d+1] = direction[d].ycos * two_inv_dyj;
//         xyzcos_dxyz[3*d+2] = direction[d].zcos * two_inv_dzk;
//      }

      for (int group = threadIdx.y; group < num_groups; group += blockDim.y){

          for (int  d = threadIdx.x; d < num_directions; d += blockDim.x){

            int gd = d + group*num_directions;

            double xcos_dxi =  direction[d].xcos * two_inv_dxi; 
            double ycos_dyj =  direction[d].ycos * two_inv_dyj;
            double zcos_dzk =  direction[d].zcos * two_inv_dzk;

            double psi_lf_z_g_d = psi_lf_z[gd];
            double psi_fr_z_g_d = psi_fr_z[gd];
            double psi_bo_z_g_d = psi_bo_z[gd];

            /* Calculate new zonal flux */
            double psi_z_g_d = (block_rhs[gd]
                + psi_lf_z_g_d * xcos_dxi
                + psi_fr_z_g_d * ycos_dyj
                + psi_bo_z_g_d * zcos_dzk)
                / (xcos_dxi + ycos_dyj + zcos_dzk + block_sigt[group]);

            block_psi[gd] = psi_z_g_d;


            /* Apply diamond-difference relationships */
            psi_lf_z[gd] = 2.0 * psi_z_g_d - psi_lf_z_g_d;
            psi_fr_z[gd] = 2.0 * psi_z_g_d - psi_fr_z_g_d;
            psi_bo_z[gd] = 2.0 * psi_z_g_d - psi_bo_z_g_d;
          }
        }
}

