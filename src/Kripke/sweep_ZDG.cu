#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "Directions.h"


#define KRESTRICT __restrict__

#define USE_PSI_HOST_MEM


//#define USE_IJK_PLANE_HOST_MEM



//#define CU_TIMING


#define MAX ((a<b)?b:a)
#define MIN ((a>b)?b:a)


//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                              \
  hipError_t e=hipGetLastError();                                     \
  if(e!=hipSuccess) {                                                  \
    printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e)); \
    exit(EXIT_FAILURE);                                                 \
  }                                                                     \
}

__global__ void  LTimes_ZDG(double *phi, double * __restrict__ psi, double * __restrict__ ell,
                                int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx);

__global__ void  LPlusTimes_ZDG(double *rhs, double * __restrict__ phi_out, double * __restrict__ ell_plus,
                                int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx);


__global__ void  sweep_over_hyperplane_ZDG(int const sliceID, int * __restrict__ offset,  int * __restrict__ ii_jj_kk_z_idx, int const num_directions, int const num_groups,
                    int const local_imax, int const local_jmax, int const local_kmax, double * __restrict__ dx, double * __restrict__ dy, double * __restrict__ dz, 
                    double * __restrict__ rhs, double *phi, double *psi, 
                    double * __restrict__ sigt, Directions * __restrict__ direction, 
                    double *i_plane, double *j_plane, double *k_plane);


int cuda_LTimes_ZDG(double *d_phi, double *h_psi, double *d_ell,
                    int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx);

int  cuda_LPlusTimes_ZDG(double *rhs, double *phi_out, double *ell_plus,
                    int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx);

int cuda_sweep_ZDG( double *rhs, double *phi, double *psi,  double *sigt, Directions *direction,
                    double *i_plane, double *j_plane, double *k_plane,
                    int *ii_jj_kk_z_idx, int *offset, int *d_offset, double *dx, double *dy, double *dz,
                    int num_zones, int num_directions, int num_groups,
                    int local_imax, int local_jmax, int local_kmax, int Nslices);


/*******************/

int cuda_LTimes_ZDG(double *d_phi, double *h_psi, double *d_ell,
                    int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx){

  
  cudaCheckError();

  dim3 threadsPerBlock(32);

  LTimes_ZDG<<<num_zones,threadsPerBlock,(num_local_groups+nidx)*sizeof(double)>>>(d_phi,h_psi,d_ell,num_zones,num_groups,num_local_directions,num_local_groups,nidx);

  cudaCheckError();


  return 0;
}


__global__ void  LTimes_ZDG(double *phi, double * __restrict__ psi, double * __restrict__ ell,
                                int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx){

      extern __shared__  double tmp_psi[];

//need more shared memory
      double *tmp_ell_d_nm = &tmp_psi[num_local_groups];


      int z = blockIdx.x;
      double *block_phi = &phi[z*num_groups*nidx];
      double *block_psi = &psi[z*num_local_groups*num_local_directions];


      for (int d = 0; d < num_local_directions; d++) {
        
        for (int group = threadIdx.x ; group < num_local_groups; group+=blockDim.x )
           tmp_psi[group] = block_psi[group + d*num_local_groups];

//use shared memory for tmp_ell_d_nm
        for (int nm_offset = threadIdx.x ; nm_offset < nidx; nm_offset += blockDim.x )
           tmp_ell_d_nm[nm_offset] =  ell[nm_offset+d*nidx];
        __syncthreads();

        for(int nm_offset = 0;nm_offset < nidx;++nm_offset){
          double ell_d_nm =  tmp_ell_d_nm[nm_offset];//  ell[nm_offset+d*nidx]; 

          for (int group = threadIdx.x ; group < num_local_groups; group+=blockDim.x )
            block_phi[group + num_groups*nm_offset] += ell_d_nm * tmp_psi[group];
                   
        }
        
    }
}

/*******************/

int  cuda_LPlusTimes_ZDG(double *d_rhs, double *h_phi_out, double *d_ell_plus,       
                    int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx){


  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float time_ms, time_s;
  double *d_phi_out;

  #ifdef CU_TIMING
  hipEventRecord(start);
  #endif

  hipMalloc((void **) &d_phi_out,num_zones*nidx * num_groups * sizeof(double));
  hipMemcpy(d_phi_out, h_phi_out,num_zones * nidx * num_groups * sizeof(double), hipMemcpyHostToDevice);
  cudaCheckError();

  #ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZDG: time to copy d_phi_out+d_ell_plus H2D: %g [s]\n",time_s);
  #endif


  #ifdef CU_TIMING
  hipEventRecord(start);
  #endif

  dim3 threadsPerBlock(32);

  LPlusTimes_ZDG<<<num_zones,threadsPerBlock,num_local_groups*sizeof(double)>>>(d_rhs,d_phi_out,d_ell_plus,num_zones,num_groups,num_local_directions,num_local_groups,nidx);

  #ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZDG: time to LPlusTimes_ZDG (GPU): %g [s]\n",time_s);
  #endif


  hipFree(d_phi_out);
  //hipFree(d_ell_plus);

  return 0;

}



__global__ void  LPlusTimes_ZDG(double *rhs, double * __restrict__ phi_out,
                                double * __restrict__ ell_plus,
                                int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx){

      extern __shared__  double tmp_rhs[];

      int z = blockIdx.x;
      double *block_rhs =  &rhs[z*num_groups*num_local_directions];
      double *block_phi_out = &phi_out[z*num_groups*nidx];

      for (int d = 0; d < num_local_directions; d++) {

        for (int group = threadIdx.x ; group < num_local_groups; group+=blockDim.x )
           tmp_rhs[group] = 0.0;

        //unroll if possible 
        for(int nm_offset = 0;nm_offset < nidx;++nm_offset){
          double ell_plus_d_n_m = ell_plus[nm_offset + d * nidx];
          for (int group = threadIdx.x ; group < num_local_groups; group+=blockDim.x ) 
            tmp_rhs[group] += ell_plus_d_n_m * block_phi_out[group + num_groups*nm_offset];
        }
        //not sure if __syncthreads  is needed, probably not;  
        __syncthreads();
        //copy from shared to global memory 
        for (int group = threadIdx.x ; group < num_local_groups; group+=blockDim.x )
            block_rhs[group + d * num_local_groups] = tmp_rhs[group];
      }
}






int cuda_sweep_ZDG( double *d_rhs, double *h_phi, double *h_psi, double *d_sigt, Directions *d_direction, 
                    double *h_i_plane, double *h_j_plane, double *h_k_plane,
                    int *d_ii_jj_kk_z_idx, int *h_offset,  int *d_offset, double *d_dx, double *d_dy, double *d_dz,
                    int num_zones, int num_directions, int num_groups,
                    int local_imax, int local_jmax, int local_kmax, int Nslices){

  size_t N;
  size_t groups_dirs;
  double *d_phi;
  //double *d_dx, *d_dy, *d_dz;
  float time_ms, time_s;
  static int  INIT_FLAG_IJK_PLANE = 0;


  N = num_zones * num_directions * num_groups;
  groups_dirs = num_directions * num_groups;


  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);


#ifdef USE_PSI_HOST_MEM
 double *d_psi = h_psi;
#else
  double *d_psi;
  #ifdef CU_TIMING
  hipEventRecord(start);
  #endif
  hipMalloc((void **) &d_psi, N*sizeof(double));
  hipMemcpy(d_psi, h_psi,   N*sizeof(double), hipMemcpyHostToDevice);

  #ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZDG: time to copy PSI H2D: %g [s]\n",time_s);
  #endif
#endif

  int i_plane_zones = local_jmax * local_kmax * groups_dirs;
  int j_plane_zones = local_imax * local_kmax * groups_dirs;
  int k_plane_zones = local_imax * local_jmax * groups_dirs;

#ifdef USE_IJK_PLANE_HOST_MEM

  double *d_i_plane = h_i_plane;
  double *d_j_plane = h_j_plane;
  double *d_k_plane = h_k_plane;

#else

  double *d_i_plane,  *d_j_plane, *d_k_plane;

  #ifdef CU_TIMING
  hipEventRecord(start);
  #endif

  hipMalloc((void **) &d_i_plane, i_plane_zones * sizeof(double));
  hipMalloc((void **) &d_j_plane, j_plane_zones * sizeof(double));
  hipMalloc((void **) &d_k_plane, k_plane_zones * sizeof(double));
  hipMemcpy(d_i_plane, h_i_plane, i_plane_zones * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_j_plane, h_j_plane, j_plane_zones * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_k_plane, h_k_plane, k_plane_zones * sizeof(double), hipMemcpyHostToDevice);
  cudaCheckError();

  #ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZDG: time to copy ijk_PLANE H2D: %g [s]\n",time_s);
  #endif

#endif


//  hipMemcpy(d_phi, h_phi,   N*sizeof(double), hipMemcpyHostToDevice);


  hipFuncSetCacheConfig(reinterpret_cast<const void*>(sweep_over_hyperplane_ZDG), hipFuncCachePreferL1);

//call cuda kernel to sweep over hyperplanes(slices)

  dim3 threadsPerBlock(32,12);

  for (int slice = 0; slice < Nslices; slice++){

#ifdef CU_TIMING    
     hipEventRecord(start); 
#endif                                            
     dim3 numBlocks = h_offset[slice+1] - h_offset[slice];
     sweep_over_hyperplane_ZDG<<<numBlocks,threadsPerBlock>>>(slice,d_offset,d_ii_jj_kk_z_idx,num_directions,num_groups,local_imax,local_jmax,local_kmax,
                                                          d_dx, d_dy, d_dz, d_rhs, d_phi, d_psi, d_sigt, d_direction, d_i_plane, d_j_plane, d_k_plane);

#ifdef CU_TIMING
    hipEventRecord(stop);                                              
    hipDeviceSynchronize();                                            
    cudaCheckError();                                                   
    float time_ms, time_s;                                              
    hipEventElapsedTime(&time_ms,start,stop);                          
    time_s=time_ms*.001;                                                
    printf("ZDG: #blocks=%d, time=%g [s]\n",h_offset[slice+1] - h_offset[slice],time_s);
#endif     
  }

  #ifdef CU_TIMING
  hipEventRecord(start);
  #endif

#ifndef USE_IJK_PLANE_HOST_MEM
    hipMemcpy(h_i_plane, d_i_plane, i_plane_zones * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_j_plane, d_j_plane, j_plane_zones * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_k_plane, d_k_plane, k_plane_zones * sizeof(double), hipMemcpyDeviceToHost);
#endif

  #ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZDG: time to copy ijk_PLANE D2H: %g [s]\n",time_s);
  #endif



#ifndef USE_PSI_HOST_MEM

#ifdef CU_TIMING
  hipEventRecord(start);
#endif

  hipMemcpy(h_psi,     d_psi, N*sizeof(double),                   hipMemcpyDeviceToHost);

#ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZDG: time to copy PSI D2H: %g [s]\n",time_s);
#endif 

  hipFree(d_psi);
#endif


//  hipFree(d_phi);

#ifndef USE_IJK_PLANE_HOST_MEM
  hipFree(d_i_plane);
  hipFree(d_j_plane);
  hipFree(d_k_plane);
#endif

  cudaCheckError();

  return 0;
}

#define I_PLANE_INDEX(j, k) ((k)*(local_jmax) + (j))
#define J_PLANE_INDEX(i, k) ((k)*(local_imax) + (i))
#define K_PLANE_INDEX(i, j) ((j)*(local_imax) + (i))


__global__ void sweep_over_hyperplane_ZDG(int const sliceID, int * __restrict__ offset, int * __restrict__ ii_jj_kk_z_idx,  int const num_directions, int const num_groups,
                    int const local_imax, int const local_jmax, int const local_kmax, double * __restrict__ dx, double * __restrict__ dy, double * __restrict__ dz, 
                    double * __restrict__ rhs, double *phi, double *psi, 
                    double * __restrict__ sigt, Directions * __restrict__ direction,
                    double *i_plane, double *j_plane, double *k_plane){

//each block will process one element 
      int element = offset[sliceID] + blockIdx.x;
      if (element > offset[sliceID+1]) return; 
      

      int i    = ii_jj_kk_z_idx[element*4];
      int j    = ii_jj_kk_z_idx[element*4+1];
      int k    = ii_jj_kk_z_idx[element*4+2];
      int z    = ii_jj_kk_z_idx[element*4+3];
      int I_P_I = I_PLANE_INDEX(j, k);
      int J_P_I = J_PLANE_INDEX(i, k);
      int K_P_I = K_PLANE_INDEX(i, j);
      double two_inv_dxi = 2.0/dx[i + 1];
      double two_inv_dyj = 2.0/dy[j + 1];
      double two_inv_dzk = 2.0/dz[k + 1];

      int dir_grp = num_directions*num_groups;

     

      double * KRESTRICT  block_rhs = &rhs[z*dir_grp];
//    double * KRESTRICT  block_phi = &phi[z*num_directions*num_groups];
      double * KRESTRICT  block_psi = &psi[z*dir_grp];
      double * KRESTRICT  block_sigt = &sigt[z*num_groups];

      double * KRESTRICT psi_lf_z_d = &i_plane[I_P_I*dir_grp]; // = i_plane.ptr(0, 0, I_P_I);
      double * KRESTRICT psi_fr_z_d = &j_plane[J_P_I*dir_grp]; // = j_plane.ptr(0, 0, J_P_I);
      double * KRESTRICT psi_bo_z_d = &k_plane[K_P_I*dir_grp]; // = k_plane.ptr(0, 0, K_P_I);


      for (int d = threadIdx.y; d < num_directions; d += blockDim.y){
          for (int group = threadIdx.x; group < num_groups; group += blockDim.x){

            double xcos = direction[d].xcos;
            double ycos = direction[d].ycos;
            double zcos = direction[d].zcos;

            int gd = group + d*num_groups;

            double xcos_dxi = xcos * two_inv_dxi;
            double ycos_dyj = ycos * two_inv_dyj;
            double zcos_dzk = zcos * two_inv_dzk;

            double psi_lf_z_d_group = psi_lf_z_d[gd];
            double psi_fr_z_d_group = psi_fr_z_d[gd];
            double psi_bo_z_d_group = psi_bo_z_d[gd];

            /* Calculate new zonal flux */
            double psi_z_d_g = (block_rhs[gd]
                + psi_lf_z_d_group * xcos_dxi
                + psi_fr_z_d_group * ycos_dyj
                + psi_bo_z_d_group * zcos_dzk)
                / (xcos_dxi + ycos_dyj + zcos_dzk + block_sigt[group]);

            block_psi[gd] = psi_z_d_g;

            /* Apply diamond-difference relationships */
            psi_lf_z_d[gd] = 2.0 * psi_z_d_g - psi_lf_z_d_group;
            psi_fr_z_d[gd] = 2.0 * psi_z_d_g - psi_fr_z_d_group;
            psi_bo_z_d[gd] = 2.0 * psi_z_d_g - psi_bo_z_d_group;
          }
      }
}





