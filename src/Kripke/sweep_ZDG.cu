#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#ifdef KRIPKE_USE_CUBLAS
/* Using updated (v2) interfaces to cublas and cusparse */
#include <hipblas.h>
#include "cu_utils.h"
#endif


#include "Directions.h"


#define KRESTRICT __restrict__

#define USE_PSI_HOST_MEM


//#define USE_IJK_PLANE_HOST_MEM

//#define CU_TIMING

#define MAX(a,b) ((a<b)?b:a)
#define MIN(a,b) ((a>b)?b:a)

//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError() {                                              \
  hipError_t e=hipGetLastError();                                     \
  if(e!=hipSuccess) {                                                  \
    printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e)); \
    exit(EXIT_FAILURE);                                                 \
  }                                                                     \
}


__global__ void  LTimes_ZDG_step1(double *phi, double * __restrict__ psi, double * __restrict__ ell,
                                int num_zones, int num_groups, int num_local_directions, int num_local_groups, 
                                int nidx, int group0);

__global__ void  LTimes_ZDG_step2(double *phi, double * __restrict__ psi, double * __restrict__ ell,
                                int num_zones, int num_groups, int num_local_directions, int num_local_groups,
                                int nidx, int group0);




__global__ void  LTimes_ZDG(double *phi, const double * __restrict__ psi, const double * __restrict__ ell,
                                int num_zones, int num_groups, int num_local_directions, int num_local_groups, 
                                int nidx, int group0);

__global__ void  LPlusTimes_ZDG(double *rhs, double * __restrict__ phi_out, const double * __restrict__ ell_plus,
                                int num_zones, int num_groups, int num_local_directions, int num_local_groups,
                                int nidx, int group0);

__global__ void scattering_ZDG(int * __restrict__ d_mixed_to_zones, int * __restrict__ d_mixed_material,
                               double * __restrict__ d_mixed_fraction, int * __restrict__ d_mixed_offset,
                               double * __restrict__ d_phi, double *d_phi_out, double * __restrict__ d_sigs0, 
                              double * __restrict__ d_sigs1, double * __restrict__ d_sigs2,
                               int * __restrict__ moment_to_coeff, int num_mixed, int num_moments, int num_groups);

__global__ void scattering_ZDG_step2(int * __restrict__ mixed_to_zones, int * __restrict__ mixed_material,
                               double * __restrict__ mixed_fraction, int * __restrict__ mixed_offset,
                               double * __restrict__ phi, double *phi_out, double * __restrict__ sigs0,
                               double * __restrict__ sigs1, double * __restrict__ sigs2,
                               int * __restrict__ moment_to_coeff, int num_mixed, int num_moments, int num_groups);

__global__ void scattering_ZDG_step3(int * __restrict__ mixed_to_zones, int * __restrict__ mixed_material,
                               double * __restrict__ mixed_fraction, int * __restrict__ mixed_offset,
                               double * __restrict__ phi, double *phi_out, double * __restrict__ sigs0,
                               double * __restrict__ sigs1, double * __restrict__ sigs2,
                               int * __restrict__ moment_to_coeff, int num_mixed, int num_moments, int num_groups);


__global__ void source_ZDG(int * __restrict__ mixed_to_zones, int * __restrict__ mixed_material, 
                               double * __restrict__ mixed_fraction, int * __restrict__ mixed_offset,
                               double *phi_out, 
                               int num_mixed, int num_moments, int num_groups);

__global__ void  sweep_over_hyperplane_ZDG(const int sliceID, int * __restrict__ offset,  int * __restrict__ ii_jj_kk_z_idx, const int  num_directions, const int  num_groups,
                    const int  local_imax, const int  local_jmax, const int local_kmax, double * __restrict__ dx, double * __restrict__ dy, double * __restrict__ dz, 
                    double * __restrict__ rhs, double *phi, double *psi, 
                    double * __restrict__ sigt, Directions * __restrict__ direction, 
                    double *i_plane, double *j_plane, double *k_plane);

__global__ void LPlusTimes_sweep_over_hyperplane_ZDG(const int sliceID, int * __restrict__ offset,  int * __restrict__ ii_jj_kk_z_idx, const int num_directions, const int num_groups,  const int num_local_groups,
                    const int nidx, const  int group0, const int local_imax, const int local_jmax, const int local_kmax, double * __restrict__ dx, double * __restrict__ dy, double * __restrict__ dz,
                    const double * __restrict__ phi_out, const double *ell_plus, double *  __restrict__ psi,
                    const double * __restrict__ sigt, Directions * __restrict__ direction,
                    double * __restrict__ i_plane, double *  __restrict__ j_plane, double *  __restrict__ k_plane);
					
__global__ void LPlusTimes_sweep_over_hyperplane_LTimes_ZDG(const int sliceID, int * __restrict__ offset,  int * __restrict__ ii_jj_kk_z_idx, 
                    const int num_directions, const int num_groups,  const int num_local_groups,
                    const int nidx, const  int group0, const int local_imax, const int local_jmax, const int local_kmax, 
                    double * __restrict__ dx, double * __restrict__ dy, double * __restrict__ dz, 
                    const double * __restrict__ phi_out, const double *ell_plus, 
                    double * __restrict__ phi, const double *ell, 
                    double *  __restrict__ psi, 
                    double * __restrict__ sigt, Directions * __restrict__ direction, 
                    double * __restrict__ i_plane, double *  __restrict__ j_plane, double *  __restrict__ k_plane);
					

int cuda_LTimes_ZDG(double *d_phi, double *h_psi, double *d_ell,
                    int num_zones, int num_groups, int num_local_directions, int num_local_groups,
                    int nidx, int group0);

#ifdef KRIPKE_USE_CUBLAS
int cuda_LPlusTimes_ZDG(double *rhs, double *phi_out, double *ell_plus, double **rhs_ptrs, double **phi_out_ptrs, double **ell_plus_ptrs,
                    int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx, int group0);
#else
int  cuda_LPlusTimes_ZDG(double *rhs, double *phi_out, double *ell_plus,
                    int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx, int group0);
#endif

int  cuda_scattering_ZDG(int *d_mixed_to_zones, int *d_mixed_material, double *d_mixed_fraction, int *d_mixed_offset,
                      double *d_phi, double *d_phi_out, double *d_sigs0,
                      double *d_sigs1, double *d_sigs2,
                      int *moment_to_coeff, int num_mixed, int num_moments, int num_groups);

int  cuda_source_ZDG(int *d_mixed_to_zones, int *d_mixed_material, double *d_mixed_fraction, int *d_mixed_offset,
                      double *d_phi_out, int num_mixed, int num_moments, int num_groups);


int cuda_sweep_ZDG( double *rhs, double *phi, double *psi,  double *sigt, Directions *direction,
                    double *i_plane, double *j_plane, double *k_plane,
                    int *ii_jj_kk_z_idx, int *offset, int *d_offset, double *dx, double *dy, double *dz,
                    int num_zones, int num_directions, int num_groups, 
                    int local_imax, int local_jmax, int local_kmax, int Nslices);

int cuda_LPlusTimes_sweep_ZDG( double *phi_out, double *ell_plus,
                    double *psi, double *sigt,  Directions *direction,
                    double *i_plane, double *j_plane, double *k_plane,
                    int *ii_jj_kk_z_idx, int *h_offset, int *d_offset, double *dx, double *dy, double *dz,
                    int num_zones, int num_directions, int num_groups,  int num_local_groups,
                    int local_imax, int local_jmax, int local_kmax, int Nslices, int nidx,  int group0);

int cuda_LPlusTimes_sweep_LTimes_ZDG( double *phi_out, double *ell_plus, 
                    double *phi, double *ell,
                    double *psi, double *sigt,  Directions *direction,
                    double *i_plane, double *j_plane, double *k_plane,
                    int *ii_jj_kk_z_idx, int *h_offset, int *d_offset, double *dx, double *dy, double *dz,
                    int num_zones, int num_directions, int num_groups,  int num_local_groups,
                    int local_imax, int local_jmax, int local_kmax, int Nslices, int nidx,  int group0);
					
/*******************/

int cuda_LTimes_ZDG(double *d_phi, double *h_psi, double *d_ell,
                    int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx, int group0){

  
  cudaCheckError();

  //dim3 threadsPerBlock(32,1); 

  #ifdef CU_TIMING
  hipEvent_t start,stop;
  float time_ms, time_s;

  int device_sync_method = hipDeviceScheduleBlockingSync; // by default we use BlockingSync
  int eventflags = ((device_sync_method == hipDeviceScheduleBlockingSync) ? hipEventBlockingSync: hipEventDefault);
  hipEventCreateWithFlags(&start, eventflags);
  hipEventCreateWithFlags(&stop, eventflags);

  hipEventRecord(start);
  #endif

//  dim3 threadsPerBlock(32,1)
//   LTimes_ZDG_step1<<<num_zones,threadsPerBlock>>>(d_phi,h_psi,d_ell,num_zones,num_groups,num_local_directions,num_local_groups,nidx);

   dim3 threadsPerBlock(32,4);
   dim3 blocksPerGrid(512);
//   LTimes_ZDG_step2<<<num_zones,threadsPerBlock>>>(d_phi,h_psi,d_ell,num_zones,num_groups,num_local_directions,num_local_groups,nidx);


  LTimes_ZDG<<<blocksPerGrid,threadsPerBlock,(num_local_groups*4)*sizeof(double)>>>(d_phi,h_psi,d_ell,num_zones,num_groups,num_local_directions,num_local_groups,nidx, group0);


  #ifdef CU_TIMING

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time_ms,start,stop);

  time_s=time_ms*.001;
  printf("ZDG: LTimes_ZDG: %g [s]\n",time_s);
  #endif



  cudaCheckError();


  return 0;
}


//LTimes_ZDG  is designed for group0 = 0; it may fail if group0 > 0


__global__ void  LTimes_ZDG_step1(double *phi, double * __restrict__ psi, double * __restrict__ ell,
                                int num_zones, int num_groups, int num_local_directions, int num_local_groups,
                                int nidx, int group0){

      int z = blockIdx.x;
      double *block_phi = &phi[z*num_groups*nidx+group0]; //  [z*num_groups*nidx + group0] ?
      double *block_psi = &psi[z*num_local_groups*num_local_directions];

      for (int d = 0; d < num_local_directions; d++) {
        
        for(int nm_offset = 0; nm_offset < nidx; nm_offset += 1){
          double ell_d_nm =  ell[nm_offset+d*nidx]; 

          for (int group = threadIdx.x ; group < num_local_groups; group+=blockDim.x )
            block_phi[group + num_groups*nm_offset] += 
                               ell_d_nm * block_psi[group + d*num_local_groups]; 
                   
        }
     }

}

__global__ void  LTimes_ZDG_step2(double *phi, double * __restrict__ psi, double * __restrict__ ell,
                                int num_zones, int num_groups, int num_local_directions, int num_local_groups,
                                int nidx, int group0){

      int z = blockIdx.x;
      double *block_phi = &phi[z*num_groups*nidx]; //  [z*num_groups*nidx + group0] ?
      double *block_psi = &psi[z*num_local_groups*num_local_directions];

      for (int d = 0; d < num_local_directions; d++) {
        
        for(int nm_offset = threadIdx.y; nm_offset < nidx; nm_offset += blockDim.y){
          double ell_d_nm =  ell[nm_offset+d*nidx]; 

          for (int group = threadIdx.x ; group < num_local_groups; group+=blockDim.x )
            block_phi[group + num_groups*nm_offset + group0] += 
                               ell_d_nm * block_psi[group + d*num_local_groups]; 
                   
        }
     }

}


__global__ void  LTimes_ZDG(double *phi, const double * __restrict__ psi, const double * __restrict__ ell,
                                int num_zones, int num_groups, int num_local_directions, int num_local_groups,
                                int nidx, int group0){

      extern __shared__  double tmp_psi[];

    for(int z = blockIdx.x; z < num_zones; z += gridDim.x){

      int gid = threadIdx.x + threadIdx.y*blockDim.x;

      double *block_phi = &phi[z*num_groups*nidx]; //  [z*num_groups*nidx + group0] ?
      const double *block_psi = &psi[z*num_local_groups*num_local_directions];

#if 1 
      int d, N_d_unrl = 4;
      double *psi_ptr[4]; //LG size must have a constant value, can not use "double *psi_ptr[N_d_unrl]" !!!
      double ell_d_nm_set[4];

      for (int i=0; i < N_d_unrl; ++i)
        psi_ptr[i] = &tmp_psi[i*num_local_groups];
      
      for (d = 0; d < num_local_directions-(N_d_unrl-1); d+=N_d_unrl) {

        //load data into shared memory
        for (int i=0; i < N_d_unrl; ++i){
          int shift = (d+i)*num_local_groups;
          for (int group = gid ; group < num_local_groups; group+= (blockDim.x*blockDim.y))
             psi_ptr[i][group] = block_psi[group + shift];
        }

        __syncthreads();

        for(int nm_offset = threadIdx.y; nm_offset < nidx; nm_offset += blockDim.y){

          for (int i=0; i < N_d_unrl; ++i)
            ell_d_nm_set[i] = ell[nm_offset+(d+i)*nidx];

          for (int group = threadIdx.x ; group < num_local_groups; group+=blockDim.x ){
            double sum = block_phi[group + num_groups*nm_offset + group0];
            for (int i=0; i < N_d_unrl; ++i) 
              sum += ell_d_nm_set[i] * psi_ptr[i][group];
            block_phi[group + num_groups*nm_offset + group0] = sum;
          }
        }
    }
    int dd = d;

    for (d = dd; d < num_local_directions; d+=1) {

        for (int group = gid ; group < num_local_groups; group+= (blockDim.x*blockDim.y))
           psi_ptr[0][group] = block_psi[group + d*num_local_groups];    //psi is read from CPU memory ,

        __syncthreads();

        for(int nm_offset = threadIdx.y; nm_offset < nidx; nm_offset += blockDim.y){
           double ell_d_nm_1 = ell[nm_offset+d*nidx];

          for (int group = threadIdx.x ; group < num_local_groups; group+=blockDim.x )
            block_phi[group + num_groups*nm_offset + group0] += (ell_d_nm_1 * psi_ptr[0][group]);
        }
    }

/*
      int group_block = 32;
      int n_group_blocks = (num_local_groups + group_block-1) /group_block;
      int group_start = 0;

    for (int gd = 0; gd < n_group_blocks; ++gd, group_start += group_block){ 

      int n_groups = MIN(group_block,num_local_groups - gd*group_block);

      for (int group = threadIdx.x ; group < n_groups; group+=blockDim.x )
        tmp_phi[group] = 0.0;

      for (int d = 0; d < num_local_directions; d++) {

//        __syncthreads();

        for (int group = threadIdx.x; group < n_groups; group+= blockDim.x)
           tmp_psi[group] = block_psi[group_start + group + d*num_local_groups];    //psi is read from CPU memory ,

//        __syncthreads();


        for(int nm_offset = 0; nm_offset < nidx; nm_offset += 1){
          double ell_d_nm =  ell[nm_offset+d*nidx];

          for (int group = threadIdx.x ; group < n_groups; group+=blockDim.x )
            block_phi[group_start + group + num_groups*nm_offset] += ell_d_nm * tmp_psi[group];     // read and write of block_phi consumes a lot of BW

        }
         
      }
    }
*/
#else

      double *tmp_ell_d_nm = &tmp_psi[num_local_groups];

      for (int d = 0; d < num_local_directions; d++) {
        
        for (int group = gid ; group < num_local_groups; group+= (blockDim.x*blockDim.y))
           tmp_psi[group] = block_psi[group + d*num_local_groups];    //psi is read from CPU memory , 

//use shared memory for tmp_ell_d_nm
        for (int nm_offset = gid ; nm_offset < nidx; nm_offset += (blockDim.x*blockDim.y) )
           tmp_ell_d_nm[nm_offset] =  ell[nm_offset+d*nidx];
        __syncthreads();


        for(int nm_offset = threadIdx.y; nm_offset < nidx; nm_offset += blockDim.y){
          double ell_d_nm =  tmp_ell_d_nm[nm_offset]; 

          for (int group = threadIdx.x ; group < num_local_groups; group+=blockDim.x )
            block_phi[group + num_groups*nm_offset] += ell_d_nm * tmp_psi[group];     // read and write of block_phi consumes a lot of BW
                   
        }
    }
#endif
  }

}

/*******************/
int  cuda_scattering_ZDG(int *d_mixed_to_zones, int *d_mixed_material, double *d_mixed_fraction, int *d_mixed_offset,
                      double *d_phi, double *d_phi_out, double *d_sigs0, double *d_sigs1, double *d_sigs2,
                      int *d_moment_to_coeff, int num_mixed, int num_moments, int num_groups){

     int dim_y = 4;
     dim3 threadsPerBlock(32,dim_y);

//     scattering_ZDG_step2<<<480,threadsPerBlock,num_groups*dim_y*sizeof(double)>>>(d_mixed_to_zones,d_mixed_material,d_mixed_fraction,d_mixed_offset,
//                                d_phi,d_phi_out,d_sigs0, d_sigs1, d_sigs2, d_moment_to_coeff,num_mixed,num_moments,num_groups);

    scattering_ZDG_step3<<<480,threadsPerBlock>>>(d_mixed_to_zones,d_mixed_material,d_mixed_fraction,d_mixed_offset,
                                d_phi,d_phi_out,d_sigs0, d_sigs1, d_sigs2, d_moment_to_coeff,num_mixed,num_moments,num_groups);


     cudaCheckError();

    return 0;
}

/*******************/


__global__ void scattering_ZDG(int * __restrict__ mixed_to_zones, int * __restrict__ mixed_material, 
                               double * __restrict__ mixed_fraction, int * __restrict__ mixed_offset,
                               double * __restrict__ phi, double *phi_out, double * __restrict__ sigs0,
                               double * __restrict__ sigs1, double * __restrict__ sigs2,
                               int * __restrict__ moment_to_coeff, int num_mixed, int num_moments, int num_groups){

   int mix_min = mixed_offset[blockIdx.x];
   int mix_max = mixed_offset[blockIdx.x+1];
   double *d_sigs[3];
   d_sigs[0] = sigs0;
   d_sigs[1] = sigs1;
   d_sigs[2] = sigs2;

   for(int mix = mix_min;mix < mix_max;++ mix){
      int zone = mixed_to_zones[mix];
      int material = mixed_material[mix];
      double fraction = mixed_fraction[mix];
      double *sigs_mat = d_sigs[material];
      double *phi_z_nm = &phi[zone*num_groups*num_moments];
      double *phi_out_z_nm = &phi_out[zone*num_groups*num_moments];

      for(int nm = threadIdx.y; nm < num_moments; nm += blockDim.y){
        // map nm to n
        int n = moment_to_coeff[nm];
        double *sigs_n_g = &sigs_mat[n*num_groups*num_groups];

        for(int g = 0;g < num_groups;++ g){
          //double *phi_out_z_gp = &phi_out[zone*num_groups*num_moments];

          for(int gp = threadIdx.x; gp < num_groups; gp += blockDim.x){
            phi_out_z_nm[gp + nm*num_groups] += sigs_n_g[gp + g*num_groups] * phi_z_nm[g + nm*num_groups] * fraction;
          }
          
        }
      }
    }
}

__global__ void scattering_ZDG_step2(int * __restrict__ mixed_to_zones, int * __restrict__ mixed_material,
                               double * __restrict__ mixed_fraction, int * __restrict__ mixed_offset,
                               double * __restrict__ phi, double *phi_out, double * __restrict__ sigs0,
                               double * __restrict__ sigs1, double * __restrict__ sigs2,
                               int * __restrict__ moment_to_coeff, int num_mixed, int num_moments, int num_groups){

   extern  __shared__ double  phi_out_z_nm_ss[]; 

   int mix_min = mixed_offset[blockIdx.x];
   int mix_max = mixed_offset[blockIdx.x+1];
   double *d_sigs[3];
   d_sigs[0] = sigs0;
   d_sigs[1] = sigs1;
   d_sigs[2] = sigs2;

   for(int mix = mix_min;mix < mix_max;++ mix){
      int zone = mixed_to_zones[mix];
      int material = mixed_material[mix];
      const double fraction = mixed_fraction[mix];
      const double * __restrict__ sigs_mat = d_sigs[material];
      const double * __restrict__ phi_z_nm = &phi[zone*num_groups*num_moments];
      double * __restrict__ phi_out_z_nm = &phi_out[zone*num_groups*num_moments];

      for(int nm = threadIdx.y; nm < num_moments; nm += blockDim.y){
        // map nm to n
        int n = moment_to_coeff[nm];
        const double * __restrict__ sigs_n_g = &sigs_mat[n*num_groups*num_groups];

        double *phi_out_z_nm_ss_y = &phi_out_z_nm_ss[num_groups*threadIdx.y]; 

        for(int gp = threadIdx.x; gp < num_groups; gp += blockDim.x)
           phi_out_z_nm_ss_y[gp] = 0.0;
 
        int shift =  nm*num_groups;
        
        for(int g = 0;g < num_groups;++ g){

          const double  scale = phi_z_nm[g + shift] * fraction; 
          for(int gp = threadIdx.x; gp < num_groups; gp += blockDim.x)
            phi_out_z_nm_ss_y[gp] += sigs_n_g[gp + g*num_groups] * scale;// phi_z_nm[g + nm*num_groups] * fraction;
          
        }
        for(int gp = threadIdx.x; gp < num_groups; gp += blockDim.x)
          phi_out_z_nm[gp + shift] += phi_out_z_nm_ss_y[gp];
      }
    }
}

__global__ void scattering_ZDG_step3(int * __restrict__ mixed_to_zones, int * __restrict__ mixed_material,
                               double * __restrict__ mixed_fraction, int * __restrict__ mixed_offset,
                               double * __restrict__ phi, double *phi_out, double * __restrict__ sigs0,
                               double * __restrict__ sigs1, double * __restrict__ sigs2,
                               int * __restrict__ moment_to_coeff, int num_mixed, int num_moments, int num_groups){

   int mix_min = mixed_offset[blockIdx.x];
   int mix_max = mixed_offset[blockIdx.x+1];
   double *d_sigs[3];
   d_sigs[0] = sigs0;
   d_sigs[1] = sigs1;
   d_sigs[2] = sigs2;

   for(int mix = mix_min;mix < mix_max;++ mix){
      int zone = mixed_to_zones[mix];
      int material = mixed_material[mix];
      const double fraction = mixed_fraction[mix];
      const double * __restrict__ sigs_mat = d_sigs[material];
      const double * __restrict__ phi_z_nm = &phi[zone*num_groups*num_moments];
      double * __restrict__ phi_out_z_nm = &phi_out[zone*num_groups*num_moments];

      for(int nm = threadIdx.y; nm < num_moments; nm += blockDim.y){
        // map nm to n
        int n = moment_to_coeff[nm];
        const double * __restrict__ sigs_n_g = &sigs_mat[n*num_groups*num_groups];
        int shift =  nm*num_groups;
        
        for (int grp_bl = 0; grp_bl < (num_groups + blockDim.x - 1 )/blockDim.x; grp_bl += 1){
          double po_z_nm = 0.0;

          int gp = grp_bl*blockDim.x + threadIdx.x;
          if (gp < num_groups){
            for(int g = 0;g < num_groups;++ g){
              const double  scale = phi_z_nm[g + shift] * fraction;
              po_z_nm  += sigs_n_g[gp + g*num_groups] * scale;// phi_z_nm[g + nm*num_groups] * fraction;
            }
          }
          phi_out_z_nm[gp + shift] += po_z_nm;
        }
      }
    }
}

int  cuda_source_ZDG(int *d_mixed_to_zones, int *d_mixed_material, double *d_mixed_fraction, int *d_mixed_offset,
                      double *d_phi_out, int num_mixed, int num_moments, int num_groups){

     dim3 threadsPerBlock(64,1);
     source_ZDG<<<480,threadsPerBlock>>>(d_mixed_to_zones,d_mixed_material,d_mixed_fraction,d_mixed_offset,
                                d_phi_out, num_mixed, num_moments, num_groups);

     cudaCheckError();

     return 0;
}

__global__ void source_ZDG(int * __restrict__ mixed_to_zones, int * __restrict__ mixed_material, 
                               double * __restrict__ mixed_fraction, int * __restrict__ mixed_offset,
                               double *phi_out, 
                               int num_mixed, int num_moments, int num_groups){

   int mix_min = mixed_offset[blockIdx.x];
   int mix_max = mixed_offset[blockIdx.x+1];
  
   for(int mix = mix_min;mix < mix_max;++ mix){

      int material = mixed_material[mix];

      if (material == 0){
        int zone = mixed_to_zones[mix];
        double fraction = mixed_fraction[mix];
        double *phi_out_z_nm0 = &phi_out[zone*num_moments*num_groups];
        for(int g = threadIdx.x;g < num_groups; g+=blockDim.x){
            phi_out_z_nm0[g] += 1.0 * fraction;
        }
      }
   }
}

#ifdef KRIPKE_USE_CUBLAS
int  cuda_LPlusTimes_ZDG(double *d_rhs, double *d_phi_out, double *d_ell_plus, double **d_rhs_ptrs, double **d_phi_out_ptrs, double **d_ell_plus_ptrs,
                    int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx, int group0){
#else
int  cuda_LPlusTimes_ZDG(double *d_rhs, double *d_phi_out, double *d_ell_plus,       
                    int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx, int group0){
#endif

//  #ifdef CU_TIMING
  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float time_ms, time_s;

  hipEventRecord(start);
//  #endif


#ifndef KRIPKE_USE_CUBLAS


//LG  adjust dim_y not to exceed the shared memory!
  int dim_y = MIN(8,num_local_directions);
  dim3 threadsPerBlock(32,dim_y);
  dim3 blocksPerGrid(256); 
  //LPlusTimes_ZDG<<<num_zones,threadsPerBlock,dim_y*num_local_groups*sizeof(double)>>>(d_rhs,d_phi_out,d_ell_plus,num_zones,num_groups,num_local_directions,num_local_groups,nidx,group0);

  LPlusTimes_ZDG<<<blocksPerGrid,threadsPerBlock>>>(d_rhs,d_phi_out,d_ell_plus,num_zones,num_groups,num_local_directions,num_local_groups,nidx,group0);



#else
  double ONE = 1.0;
  double ZERO = 0.0;

  hipblasOperation_t transa = HIPBLAS_OP_N;
  hipblasOperation_t transb = HIPBLAS_OP_N;

  hipblasHandle_t handle;
  handle = get_cublasHandle();

  #ifdef CU_TIMING
  hipEvent_t start_dgemm,stop_dgemm;
  hipEventCreate(&start_dgemm);
  hipEventCreate(&stop_dgemm);
  hipEventRecord(start_dgemm);
  #endif

  hipblasDgemmBatched(handle,
                     transa, transb,
                     num_local_groups, num_local_directions, nidx, 
                     &ONE,
                     (const double **) d_phi_out_ptrs, num_groups,
                     (const double **) d_ell_plus_ptrs, nidx,
                     &ONE,
                     d_rhs_ptrs, num_local_groups, num_zones);

  #ifdef CU_TIMING
  hipEventRecord(stop_dgemm);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start_dgemm,stop_dgemm);
  time_s=time_ms*.001;
  printf("ZDG:  LPlusTimes_ZDG - hipblasDgemmBatched : %g [s]  \n",time_s);
  #endif

#endif


  #ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZDG: time to LPlusTimes_ZDG (GPU): %g [s]\n",time_s);
  #endif

  return 0;

}



__global__ void  LPlusTimes_ZDG(double *rhs, double * __restrict__ phi_out,
                                const double * __restrict__ ell_plus,
                                int num_zones, int num_groups, int num_local_directions, int num_local_groups, int nidx, int group0){

    //extern __shared__  double tmp_rhs[];


    for(int z = blockIdx.x; z < num_zones; z += gridDim.x){
 
      double * KRESTRICT block_rhs =  &rhs[z*num_local_groups*num_local_directions];
      const double * KRESTRICT  block_phi_out = &phi_out[z*num_groups*nidx + group0];


#if 1

      for (int d = threadIdx.y; d < num_local_directions; d += blockDim.y) {
          for (int group = threadIdx.x ; group < num_local_groups; group+=blockDim.x ){
            double r0 = 0.0;
            for(int nm_offset = 0;nm_offset < nidx;++nm_offset){
                double ell_plus_d_n_m = ell_plus[nm_offset + d * nidx];
                r0  += ell_plus_d_n_m * block_phi_out[group + num_groups*nm_offset];
            }
            block_rhs[group + d * num_local_groups] = r0;
          }
      }

#else
      //requires shared memory
      
      for (int d = threadIdx.y; d < num_local_directions; d += blockDim.y) {
           double *tmp_rhs_d = &tmp_rhs[threadIdx.y*num_local_groups];


        for (int group = threadIdx.x ; group < num_local_groups; group+=blockDim.x )
           tmp_rhs_d[group] = 0.0;

        //unroll if possible
        for(int nm_offset = 0;nm_offset < nidx;++nm_offset){
          double ell_plus_d_n_m = ell_plus[nm_offset + d * nidx];
          for (int group = threadIdx.x ; group < num_local_groups; group+=blockDim.x )
            tmp_rhs_d[group] += ell_plus_d_n_m * block_phi_out[group + num_groups*nm_offset];// + group0];
        }
        //copy from shared to global memory
        for (int group = threadIdx.x ; group < num_local_groups; group+=blockDim.x )
            block_rhs[group + d * num_local_groups] = tmp_rhs_d[group];
      }
#endif
    }
 
#if 0

      //requires shared memory

      for (int d = 0; d < num_local_directions; d++) {

        for (int group = threadIdx.x ; group < num_local_groups; group+=blockDim.x )
           tmp_rhs[group] = 0.0;

        //unroll if possible 
        for(int nm_offset = 0;nm_offset < nidx;++nm_offset){
          double ell_plus_d_n_m = ell_plus[nm_offset + d * nidx];
          for (int group = threadIdx.x ; group < num_local_groups; group+=blockDim.x ) 
            tmp_rhs[group] += ell_plus_d_n_m * block_phi_out[group + num_groups*nm_offset];
        }
        //not sure if __syncthreads  is needed, probably not;  
        __syncthreads();
        //copy from shared to global memory 
        for (int group = threadIdx.x ; group < num_local_groups; group+=blockDim.x )
            block_rhs[group + d * num_local_groups] = tmp_rhs[group];
      }
#endif
}






int cuda_sweep_ZDG( double *d_rhs, double *h_phi, double *h_psi, double *d_sigt, Directions *d_direction, 
                    double *h_i_plane, double *h_j_plane, double *h_k_plane,
                    int *d_ii_jj_kk_z_idx, int *h_offset,  int *d_offset, double *d_dx, double *d_dy, double *d_dz,
                    int num_zones, int num_directions, int num_groups,
                    int local_imax, int local_jmax, int local_kmax, int Nslices){

  size_t N;
  size_t groups_dirs;
  double *d_phi;
  //double *d_dx, *d_dy, *d_dz;
  float time_ms, time_s;
  static int  INIT_FLAG_IJK_PLANE = 0;


  N = num_zones * num_directions * num_groups;
  groups_dirs = num_directions * num_groups;


  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);


#ifdef USE_PSI_HOST_MEM
 double *d_psi = h_psi;
#else
  double *d_psi;
  #ifdef CU_TIMING
  hipEventRecord(start);
  #endif
  hipMalloc((void **) &d_psi, N*sizeof(double));
  hipMemcpy(d_psi, h_psi,   N*sizeof(double), hipMemcpyHostToDevice);

  #ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZDG: time to copy PSI H2D: %g [s]\n",time_s);
  #endif
#endif

  int i_plane_zones = local_jmax * local_kmax * groups_dirs;
  int j_plane_zones = local_imax * local_kmax * groups_dirs;
  int k_plane_zones = local_imax * local_jmax * groups_dirs;

#ifdef USE_IJK_PLANE_HOST_MEM

  double *d_i_plane = h_i_plane;
  double *d_j_plane = h_j_plane;
  double *d_k_plane = h_k_plane;

#else

  double *d_i_plane,  *d_j_plane, *d_k_plane;

  #ifdef CU_TIMING
  hipEventRecord(start);
  #endif

  hipMalloc((void **) &d_i_plane, (i_plane_zones + j_plane_zones + k_plane_zones) * sizeof(double));
  d_j_plane = d_i_plane + i_plane_zones;
  d_k_plane = d_i_plane + i_plane_zones + j_plane_zones;
  hipMemcpy(d_i_plane, h_i_plane, i_plane_zones * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_j_plane, h_j_plane, j_plane_zones * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_k_plane, h_k_plane, k_plane_zones * sizeof(double), hipMemcpyHostToDevice);
  cudaCheckError();

  #ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZDG: time to copy ijk_PLANE H2D: %g [s]\n",time_s);
  #endif

#endif



//  hipFuncSetCacheConfig(reinterpret_cast<const void*>(sweep_over_hyperplane_ZDG), hipFuncCachePreferL1);

//call cuda kernel to sweep over hyperplanes(slices)

 
  int dim_y, griddim_y=1;  
  dim_y = min(6,num_directions); //6
  dim3 threadsPerBlock(32,dim_y);

#ifdef CU_TIMING    
     hipEventRecord(start); 
#endif             
  for (int slice = 0; slice < Nslices; slice++){     
     int nzones = h_offset[slice+1] - h_offset[slice];

     if (nzones < 45) { griddim_y = 6;}//   dim_y = min(4,num_directions);}   //4
     else             { griddim_y = 2;}//   dim_y = min(4,num_directions);}   //2

     dim3 numBlocks(nzones,griddim_y);
                          
     sweep_over_hyperplane_ZDG<<<numBlocks,threadsPerBlock>>>(slice,d_offset,d_ii_jj_kk_z_idx,num_directions,num_groups,local_imax,local_jmax,local_kmax,
                                                          d_dx, d_dy, d_dz, d_rhs, d_phi, d_psi, d_sigt, d_direction, d_i_plane, d_j_plane, d_k_plane);
  } 
#ifdef CU_TIMING
    hipEventRecord(stop);                                              
    hipDeviceSynchronize();                                            
    cudaCheckError();                                                   
                                                  
    hipEventElapsedTime(&time_ms,start,stop);                          
    time_s=time_ms*.001;               
    printf("ZDG: sweep_time= %g [s]\n", time_s);                                 
  //  printf("ZDG: #blocks=%d, time=%g [s]\n",h_offset[slice+1] - h_offset[slice],time_s);
#endif     
  

  #ifdef CU_TIMING
  hipEventRecord(start);
  #endif

#ifndef USE_IJK_PLANE_HOST_MEM
    hipMemcpy(h_i_plane, d_i_plane, i_plane_zones * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_j_plane, d_j_plane, j_plane_zones * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_k_plane, d_k_plane, k_plane_zones * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_i_plane);
#endif


  #ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZDG: time to copy ijk_PLANE D2H: %g [s]\n",time_s);
  #endif



#ifndef USE_PSI_HOST_MEM

#ifdef CU_TIMING
  hipEventRecord(start);
#endif

  hipMemcpy(h_psi,     d_psi, N*sizeof(double),                   hipMemcpyDeviceToHost);

#ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZDG: time to copy PSI D2H: %g [s]\n",time_s);
#endif 

  hipFree(d_psi);
#endif

  cudaCheckError();

  return 0;
}

#define I_PLANE_INDEX(j, k) ((k)*(local_jmax) + (j))
#define J_PLANE_INDEX(i, k) ((k)*(local_imax) + (i))
#define K_PLANE_INDEX(i, j) ((j)*(local_imax) + (i))


__global__ void sweep_over_hyperplane_ZDG(int const sliceID, int * __restrict__ offset, int * __restrict__ ii_jj_kk_z_idx,  int const num_directions, int const num_groups,
                    int const local_imax, int const local_jmax, int const local_kmax, double * __restrict__ dx, double * __restrict__ dy, double * __restrict__ dz, 
                    double * __restrict__ rhs, double *phi, double *psi, 
                    double * __restrict__ sigt, Directions * __restrict__ direction,
                    double *i_plane, double *j_plane, double *k_plane){

//each block will process one element 
      int element = offset[sliceID] + blockIdx.x;
      if (element > offset[sliceID+1]) return; 
      

      int i    = ii_jj_kk_z_idx[element*4];
      int j    = ii_jj_kk_z_idx[element*4+1];
      int k    = ii_jj_kk_z_idx[element*4+2];
      int z    = ii_jj_kk_z_idx[element*4+3];
      int I_P_I = I_PLANE_INDEX(j, k);
      int J_P_I = J_PLANE_INDEX(i, k);
      int K_P_I = K_PLANE_INDEX(i, j);
      double two_inv_dxi = 2.0/dx[i + 1];
      double two_inv_dyj = 2.0/dy[j + 1];
      double two_inv_dzk = 2.0/dz[k + 1];

      int dir_grp = num_directions*num_groups;

      double * KRESTRICT  block_rhs = &rhs[z*dir_grp];
//    double * KRESTRICT  block_phi = &phi[z*dir_grp];
      double * KRESTRICT  block_psi = &psi[z*dir_grp];
      double * KRESTRICT  block_sigt = &sigt[z*num_groups];

      double * KRESTRICT psi_lf_z_d = &i_plane[I_P_I*dir_grp]; // = i_plane.ptr(0, 0, I_P_I);
      double * KRESTRICT psi_fr_z_d = &j_plane[J_P_I*dir_grp]; // = j_plane.ptr(0, 0, J_P_I);
      double * KRESTRICT psi_bo_z_d = &k_plane[K_P_I*dir_grp]; // = k_plane.ptr(0, 0, K_P_I);

      int chunk = (num_directions + gridDim.y - 1) / gridDim.y;
      int dstart =   chunk*blockIdx.y + threadIdx.y;
      int dend   = MIN( (blockIdx.y+1)*chunk, num_directions);

     // for (int d = threadIdx.y; d < num_directions; d += blockDim.y){

      for (int d = dstart; d < dend; d += blockDim.y){
          for (int group = threadIdx.x; group < num_groups; group += blockDim.x){

            double xcos = direction[d].xcos;
            double ycos = direction[d].ycos;
            double zcos = direction[d].zcos;

            int gd = group + d*num_groups;

            double xcos_dxi = xcos * two_inv_dxi;
            double ycos_dyj = ycos * two_inv_dyj;
            double zcos_dzk = zcos * two_inv_dzk;

            double psi_lf_z_d_group = psi_lf_z_d[gd];
            double psi_fr_z_d_group = psi_fr_z_d[gd];
            double psi_bo_z_d_group = psi_bo_z_d[gd];

            /* Calculate new zonal flux */
            double psi_z_d_g = (block_rhs[gd]
                + psi_lf_z_d_group * xcos_dxi
                + psi_fr_z_d_group * ycos_dyj
                + psi_bo_z_d_group * zcos_dzk)
                / (xcos_dxi + ycos_dyj + zcos_dzk + block_sigt[group]);

            block_psi[gd] = psi_z_d_g;

            /* Apply diamond-difference relationships */
            psi_lf_z_d[gd] = 2.0 * psi_z_d_g - psi_lf_z_d_group;
            psi_fr_z_d[gd] = 2.0 * psi_z_d_g - psi_fr_z_d_group;
            psi_bo_z_d[gd] = 2.0 * psi_z_d_g - psi_bo_z_d_group;
          }
      }
}



int cuda_LPlusTimes_sweep_ZDG( double *d_phi_out, double *d_ell_plus,
                    double *h_psi, double *d_sigt,  Directions *d_direction,
                    double *h_i_plane, double *h_j_plane, double *h_k_plane,
                    int *d_ii_jj_kk_z_idx, int *h_offset, int *d_offset, 
                    double *d_dx, double *d_dy, double *d_dz,
                    int num_zones, int num_directions, int num_groups, int num_local_groups,
                    int local_imax, int local_jmax, int local_kmax, int Nslices, int nidx, int group0){

  size_t N;
  size_t groups_dirs;
  float time_ms, time_s;
  static int  INIT_FLAG_IJK_PLANE = 0;

  N = num_zones * num_directions * num_local_groups;
  groups_dirs = num_directions * num_local_groups;

  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

#ifdef USE_PSI_HOST_MEM
 double *d_psi = h_psi;
#else
  double *d_psi;
  #ifdef CU_TIMING
  hipEventRecord(start);
  #endif
  hipMalloc((void **) &d_psi, N*sizeof(double));
  hipMemcpy(d_psi, h_psi,   N*sizeof(double), hipMemcpyHostToDevice);

  #ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZDG: time to copy PSI H2D: %g [s]\n",time_s);
  #endif
#endif

  int i_plane_zones = local_jmax * local_kmax * groups_dirs;
  int j_plane_zones = local_imax * local_kmax * groups_dirs;
  int k_plane_zones = local_imax * local_jmax * groups_dirs;

#ifdef USE_IJK_PLANE_HOST_MEM

  double *d_i_plane = h_i_plane;
  double *d_j_plane = h_j_plane;
  double *d_k_plane = h_k_plane;

#else

  double *d_i_plane,  *d_j_plane, *d_k_plane;

  #ifdef CU_TIMING
  hipEventRecord(start);
  #endif

  hipMalloc((void **) &d_i_plane, (i_plane_zones + j_plane_zones + k_plane_zones) * sizeof(double));
  d_j_plane = d_i_plane + i_plane_zones;
  d_k_plane = d_i_plane + i_plane_zones + j_plane_zones; 
  hipMemcpy(d_i_plane, h_i_plane, i_plane_zones * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_j_plane, h_j_plane, j_plane_zones * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_k_plane, h_k_plane, k_plane_zones * sizeof(double), hipMemcpyHostToDevice);



  cudaCheckError();

  #ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZDG: time to copy ijk_PLANE H2D: %g [s]\n",time_s);
  #endif

#endif



//  hipFuncSetCacheConfig(reinterpret_cast<const void*>(LPlusTimes_sweep_over_hyperplane_ZDG), hipFuncCachePreferL1);

//call cuda kernel to sweep over hyperplanes(slices)


  int dim_y, griddim_y;  
  dim_y = min(4,num_directions); //6
  dim3 threadsPerBlock(32,dim_y);


#ifdef CU_TIMING    
     hipEventRecord(start); 
#endif             
  
  for (int slice = 0; slice < Nslices; slice++){        
     int nzones = h_offset[slice+1] - h_offset[slice];
     if (nzones < 45) { griddim_y = 6;}//   dim_y = min(4,num_directions);}   //4
     else             { griddim_y = 2;}//   dim_y = min(4,num_directions);}   //2
  
//     if (nzones < 15)      { griddim_y = 12;  dim_y = min(2,num_directions);}   //4
//     else if (nzones < 60) { griddim_y = 6;  dim_y = min(4,num_directions);}   //4
//     else if (nzones < 90) { griddim_y = 2;  dim_y = min(8,num_directions);}   //4
//     else                  { griddim_y = 1;  dim_y = min(8,num_directions);}   //2

     dim3 numBlocks(nzones,griddim_y);

     LPlusTimes_sweep_over_hyperplane_ZDG<<<numBlocks,threadsPerBlock,num_local_groups*dim_y*sizeof(double)>>>(slice,d_offset,d_ii_jj_kk_z_idx,num_directions,num_groups,num_local_groups,nidx,group0,local_imax,local_jmax,local_kmax,
                                                          d_dx, d_dy, d_dz, d_phi_out, d_ell_plus, d_psi, d_sigt, d_direction, d_i_plane, d_j_plane, d_k_plane);
  } 
#ifdef CU_TIMING
    hipEventRecord(stop);                                              
    hipDeviceSynchronize();                                            
    cudaCheckError();                                                   
                                                  
    hipEventElapsedTime(&time_ms,start,stop);                          
    time_s=time_ms*.001;               
    printf("ZDG: sweep_time= %g [s]\n", time_s);                                 
  //  printf("ZDG: #blocks=%d, time=%g [s]\n",h_offset[slice+1] - h_offset[slice],time_s);
#endif     
  

  #ifdef CU_TIMING
  hipEventRecord(start);
  #endif

#ifndef USE_IJK_PLANE_HOST_MEM
    hipMemcpy(h_i_plane, d_i_plane, i_plane_zones * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_j_plane, d_j_plane, j_plane_zones * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_k_plane, d_k_plane, k_plane_zones * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_i_plane);
#endif

  #ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZDG: time to copy ijk_PLANE D2H: %g [s]\n",time_s);
  #endif



#ifndef USE_PSI_HOST_MEM

#ifdef CU_TIMING
  hipEventRecord(start);
#endif

  hipMemcpy(h_psi,     d_psi, N*sizeof(double),                   hipMemcpyDeviceToHost);

#ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZDG: time to copy PSI D2H: %g [s]\n",time_s);
#endif 

  hipFree(d_psi);
#endif


  cudaCheckError();

  return 0;
}


__global__ void LPlusTimes_sweep_over_hyperplane_ZDG(const int sliceID, int * __restrict__ offset,  int * __restrict__ ii_jj_kk_z_idx, const int num_directions, const int num_groups,  const int num_local_groups,
                    const int nidx, const  int group0, const int local_imax, const int local_jmax, const int local_kmax, double * __restrict__ dx, double * __restrict__ dy, double * __restrict__ dz, 
                    const double * __restrict__ phi_out, const double *ell_plus, double *  __restrict__ psi, 
                    const double * __restrict__ sigt, Directions * __restrict__ direction, 
                    double * __restrict__ i_plane, double *  __restrict__ j_plane, double *  __restrict__ k_plane){

      extern __shared__  double tmp_rhs[];
					
//each block will process one element 
      int element = offset[sliceID] + blockIdx.x;
      if (element > offset[sliceID+1]) return; 
      
      int i    = ii_jj_kk_z_idx[element*4];
      int j    = ii_jj_kk_z_idx[element*4+1];
      int k    = ii_jj_kk_z_idx[element*4+2];
      int z    = ii_jj_kk_z_idx[element*4+3];
      int I_P_I = I_PLANE_INDEX(j, k);
      int J_P_I = J_PLANE_INDEX(i, k);
      int K_P_I = K_PLANE_INDEX(i, j);
      double two_inv_dxi = 2.0/dx[i + 1];
      double two_inv_dyj = 2.0/dy[j + 1];
      double two_inv_dzk = 2.0/dz[k + 1];

      int dir_grp = num_directions*num_local_groups;

      const double * KRESTRICT  block_phi_out = &phi_out[z*num_groups*nidx + group0];
      double * KRESTRICT  block_psi = &psi[z*dir_grp];
      const double * KRESTRICT  block_sigt = &sigt[z*num_local_groups];

      double * KRESTRICT psi_lf_z_d = &i_plane[I_P_I*dir_grp]; // = i_plane.ptr(0, 0, I_P_I);
      double * KRESTRICT psi_fr_z_d = &j_plane[J_P_I*dir_grp]; // = j_plane.ptr(0, 0, J_P_I);
      double * KRESTRICT psi_bo_z_d = &k_plane[K_P_I*dir_grp]; // = k_plane.ptr(0, 0, K_P_I);

      double * KRESTRICT tmp_rhs_d = &tmp_rhs[threadIdx.y * num_local_groups];

      int chunk = (num_directions + gridDim.y - 1) / gridDim.y;
      int dstart =   chunk*blockIdx.y + threadIdx.y;
      int dend   = MIN( (blockIdx.y+1)*chunk, num_directions);

      for (int d = dstart; d < dend; d += blockDim.y){
 	 
          const double xcos = direction[d].xcos;
          const double ycos = direction[d].ycos;
          const double zcos = direction[d].zcos;

          for (int group = threadIdx.x ; group < num_local_groups; group+=blockDim.x ){
            double r0 = 0.0;
            for(int nm_offset = 0;nm_offset < nidx;++nm_offset){
              const double ell_plus_d_n_m = ell_plus[nm_offset + d * nidx];
              r0 += ell_plus_d_n_m * block_phi_out[group + num_groups*nm_offset];
            }
            tmp_rhs_d[group] = r0;
          }

          for (int group = threadIdx.x; group < num_local_groups; group += blockDim.x){

            int gd = group + d*num_local_groups;

            double psi_lf_z_d_group = psi_lf_z_d[gd];
            double psi_fr_z_d_group = psi_fr_z_d[gd];
            double psi_bo_z_d_group = psi_bo_z_d[gd];

            double xcos_dxi = xcos * two_inv_dxi;
            double ycos_dyj = ycos * two_inv_dyj;
            double zcos_dzk = zcos * two_inv_dzk;

            /* Calculate new zonal flux */
            double psi_z_d_g = (tmp_rhs_d[group]
                + psi_lf_z_d_group * xcos_dxi
                + psi_fr_z_d_group * ycos_dyj
                + psi_bo_z_d_group * zcos_dzk)
                / (xcos_dxi + ycos_dyj + zcos_dzk + block_sigt[group]);

            block_psi[gd] = psi_z_d_g;

            /* Apply diamond-difference relationships */
            psi_lf_z_d[gd] = 2.0 * psi_z_d_g - psi_lf_z_d_group;
            psi_fr_z_d[gd] = 2.0 * psi_z_d_g - psi_fr_z_d_group;
            psi_bo_z_d[gd] = 2.0 * psi_z_d_g - psi_bo_z_d_group;
          }
         
/* test  */
#if 0
assume tmp_rhs_d[group] = block_psi[gd] is in the GPU memory
      
           int GTID = threadIDx.x + blockDim.x*threadID.y;

           if ( (group0 == 0)  && (d/blockDim.y == 0) )
               for (int group = GTID ; group < num_groups*nidx; group+=(blockDim.x *blockDim.y) block_phi[grpup] = 0.0;
           __syncthreads();

           for (int dd = 0; dd < blockDim.y; ++dd){
              int dir = dd + d/blockDim.y;

              for(int nm_offset = 0; nm_offset < nidx; nm_offset += 1){
                double ell_d_nm =  ell[nm_offset+dir*nidx]; 
                for (int group = GTID ; group < num_local_groups; group+=(blockDim.x *blockDim.y){
                   block_phi[group + num_groups*nm_offset] += tmp_rhs[group + dd*num_local_groups] * ell[nm_offset+dir*nidx];  
                }
              } 
           }   

/*  end test */
#endif       
       } //endof "for (int d = threadIdx.y; d < num_directions; d += blockDim.y){" 


}

int cuda_LPlusTimes_sweep_LTimes_ZDG( double *d_phi_out, double *d_ell_plus,
                    double *d_phi, double *d_ell,
                    double *h_psi, double *d_sigt,  Directions *d_direction,
                    double *h_i_plane, double *h_j_plane, double *h_k_plane,
                    int *d_ii_jj_kk_z_idx, int *h_offset, int *d_offset, 
                    double *d_dx, double *d_dy, double *d_dz,
                    int num_zones, int num_directions, int num_groups, int num_local_groups,
                    int local_imax, int local_jmax, int local_kmax, int Nslices, int nidx, int group0){

  size_t N;
  size_t groups_dirs;
  float time_ms, time_s;
  static int  INIT_FLAG_IJK_PLANE = 0;

  N = num_zones * num_directions * num_local_groups;
  groups_dirs = num_directions * num_local_groups;

  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

#ifdef USE_PSI_HOST_MEM
 double *d_psi = h_psi;
#else
  double *d_psi;
  #ifdef CU_TIMING
  hipEventRecord(start);
  #endif
  hipMalloc((void **) &d_psi, N*sizeof(double));
  hipMemcpy(d_psi, h_psi,   N*sizeof(double), hipMemcpyHostToDevice);

  #ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZDG: time to copy PSI H2D: %g [s]\n",time_s);
  #endif
#endif

  int i_plane_zones = local_jmax * local_kmax * groups_dirs;
  int j_plane_zones = local_imax * local_kmax * groups_dirs;
  int k_plane_zones = local_imax * local_jmax * groups_dirs;

#ifdef USE_IJK_PLANE_HOST_MEM

  double *d_i_plane = h_i_plane;
  double *d_j_plane = h_j_plane;
  double *d_k_plane = h_k_plane;

#else

  double *d_i_plane,  *d_j_plane, *d_k_plane;

  #ifdef CU_TIMING
  hipEventRecord(start);
  #endif

  hipMalloc((void **) &d_i_plane, i_plane_zones * sizeof(double));
  hipMalloc((void **) &d_j_plane, j_plane_zones * sizeof(double));
  hipMalloc((void **) &d_k_plane, k_plane_zones * sizeof(double));
  hipMemcpy(d_i_plane, h_i_plane, i_plane_zones * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_j_plane, h_j_plane, j_plane_zones * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_k_plane, h_k_plane, k_plane_zones * sizeof(double), hipMemcpyHostToDevice);
  cudaCheckError();

  #ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZDG: time to copy ijk_PLANE H2D: %g [s]\n",time_s);
  #endif

#endif



//  hipFuncSetCacheConfig(reinterpret_cast<const void*>(LPlusTimes_sweep_over_hyperplane_ZDG), hipFuncCachePreferL1);

//call cuda kernel to sweep over hyperplanes(slices)

  int dim_y = min(12,num_directions); 
  dim3 threadsPerBlock(32,dim_y);


#ifdef CU_TIMING    
     hipEventRecord(start); 
#endif             
  for (int slice = 0; slice < Nslices; slice++){                               
     dim3 numBlocks = h_offset[slice+1] - h_offset[slice];
     LPlusTimes_sweep_over_hyperplane_LTimes_ZDG<<<numBlocks,threadsPerBlock,num_local_groups*(1+dim_y)*sizeof(double)>>>(slice,d_offset,d_ii_jj_kk_z_idx,num_directions,num_groups,num_local_groups,nidx,group0,local_imax,local_jmax,local_kmax,
                                                          d_dx, d_dy, d_dz, d_phi_out, d_ell_plus, d_phi, d_ell, d_psi, d_sigt, d_direction, d_i_plane, d_j_plane, d_k_plane);
  } 
#ifdef CU_TIMING
    hipEventRecord(stop);                                              
    hipDeviceSynchronize();                                            
    cudaCheckError();                                                   
                                                  
    hipEventElapsedTime(&time_ms,start,stop);                          
    time_s=time_ms*.001;               
    printf("ZDG: sweep_time= %g [s]\n", time_s);                                 
  //  printf("ZDG: #blocks=%d, time=%g [s]\n",h_offset[slice+1] - h_offset[slice],time_s);
#endif     
  

  #ifdef CU_TIMING
  hipEventRecord(start);
  #endif

#ifndef USE_IJK_PLANE_HOST_MEM
    hipMemcpy(h_i_plane, d_i_plane, i_plane_zones * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_j_plane, d_j_plane, j_plane_zones * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_k_plane, d_k_plane, k_plane_zones * sizeof(double), hipMemcpyDeviceToHost);
#endif

  #ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZDG: time to copy ijk_PLANE D2H: %g [s]\n",time_s);
  #endif



#ifndef USE_PSI_HOST_MEM

#ifdef CU_TIMING
  hipEventRecord(start);
#endif

  hipMemcpy(h_psi,     d_psi, N*sizeof(double),                   hipMemcpyDeviceToHost);

#ifdef CU_TIMING
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();
  hipEventElapsedTime(&time_ms,start,stop);
  time_s=time_ms*.001;
  printf("ZDG: time to copy PSI D2H: %g [s]\n",time_s);
#endif 

  hipFree(d_psi);
#endif


#ifndef USE_IJK_PLANE_HOST_MEM
  hipFree(d_i_plane);
  hipFree(d_j_plane);
  hipFree(d_k_plane);
#endif

  cudaCheckError();

  return 0;
}




__global__ void LPlusTimes_sweep_over_hyperplane_LTimes_ZDG(const int sliceID, int * __restrict__ offset,  int * __restrict__ ii_jj_kk_z_idx, 
                    const int num_directions, const int num_groups,  const int num_local_groups,
                    const int nidx, const  int group0, const int local_imax, const int local_jmax, const int local_kmax, 
                    double * __restrict__ dx, double * __restrict__ dy, double * __restrict__ dz, 
                    const double * __restrict__ phi_out, const double *ell_plus, 
                    double * __restrict__ phi, const double *ell, 
                    double *  __restrict__ psi, 
                    double * __restrict__ sigt, Directions * __restrict__ direction, 
                    double * __restrict__ i_plane, double *  __restrict__ j_plane, double *  __restrict__ k_plane){

      extern __shared__  double tmp_rhs[];
					
//each block will process one element 
      int element = offset[sliceID] + blockIdx.x;
      int GTID = threadIdx.x + blockDim.x*threadIdx.y;


      if (element > offset[sliceID+1]) return; 
      
      int i    = ii_jj_kk_z_idx[element*4];
      int j    = ii_jj_kk_z_idx[element*4+1];
      int k    = ii_jj_kk_z_idx[element*4+2];
      int z    = ii_jj_kk_z_idx[element*4+3];
      int K_P_I = K_PLANE_INDEX(i, j);
      int I_P_I = I_PLANE_INDEX(j, k);
      int J_P_I = J_PLANE_INDEX(i, k);
      double two_inv_dxi = 2.0/dx[i + 1];
      double two_inv_dyj = 2.0/dy[j + 1];
      double two_inv_dzk = 2.0/dz[k + 1];

      int dir_grp = num_directions*num_local_groups;

      const double * KRESTRICT  block_phi_out = &phi_out[z*num_groups*nidx + group0];
      double * KRESTRICT  block_phi = &phi[z*num_groups*nidx + group0];

      double * KRESTRICT  block_psi = &psi[z*dir_grp];
      double * KRESTRICT  block_sigt = &sigt[z*num_local_groups];

      double * KRESTRICT psi_lf_z_d = &i_plane[I_P_I*dir_grp]; // = i_plane.ptr(0, 0, I_P_I);
      double * KRESTRICT psi_fr_z_d = &j_plane[J_P_I*dir_grp]; // = j_plane.ptr(0, 0, J_P_I);
      double * KRESTRICT psi_bo_z_d = &k_plane[K_P_I*dir_grp]; // = k_plane.ptr(0, 0, K_P_I);

      double * KRESTRICT tmp_rhs_d = &tmp_rhs[threadIdx.y * num_local_groups];

      for (int d = threadIdx.y; d < num_directions; d += blockDim.y){
 	  
          for (int group = threadIdx.x ; group < num_local_groups; group+=blockDim.x )
             tmp_rhs_d[group] = 0.0;
      
 
          for(int nm_offset = 0;nm_offset < nidx;++nm_offset){
            double ell_plus_d_n_m = ell_plus[nm_offset + d * nidx];
            for (int group = threadIdx.x ; group < num_local_groups; group+=blockDim.x ) 
              tmp_rhs_d[group] += ell_plus_d_n_m * block_phi_out[group + num_groups*nm_offset];
          }
        
          for (int group = threadIdx.x; group < num_local_groups; group += blockDim.x){

            double xcos = direction[d].xcos;
            double ycos = direction[d].ycos;
            double zcos = direction[d].zcos;

            int gd = group + d*num_local_groups;

            double xcos_dxi = xcos * two_inv_dxi;
            double ycos_dyj = ycos * two_inv_dyj;
            double zcos_dzk = zcos * two_inv_dzk;

            double psi_lf_z_d_group = psi_lf_z_d[gd];
            double psi_fr_z_d_group = psi_fr_z_d[gd];
            double psi_bo_z_d_group = psi_bo_z_d[gd];

            /* Calculate new zonal flux */
            double psi_z_d_g = (tmp_rhs_d[group]
                + psi_lf_z_d_group * xcos_dxi
                + psi_fr_z_d_group * ycos_dyj
                + psi_bo_z_d_group * zcos_dzk)
                / (xcos_dxi + ycos_dyj + zcos_dzk + block_sigt[group]);

            block_psi[gd] = psi_z_d_g;
            tmp_rhs_d[group] = psi_z_d_g;


            /* Apply diamond-difference relationships */
            psi_lf_z_d[gd] = 2.0 * psi_z_d_g - psi_lf_z_d_group;
            psi_fr_z_d[gd] = 2.0 * psi_z_d_g - psi_fr_z_d_group;
            psi_bo_z_d[gd] = 2.0 * psi_z_d_g - psi_bo_z_d_group;
          }
         

//LG problem if num_directions%blockDim.y != 0

#if 1
          double * KRESTRICT tmp_block_phi = &tmp_rhs[blockDim.y * num_local_groups];
          int d_shift = (d/blockDim.y)*blockDim.y;
          int dd_max = MIN(blockDim.y,num_directions-d_shift); 
           __syncthreads();


          for(int nm_offset = 0; nm_offset < nidx; nm_offset += 1){

            for (int group = GTID ; group < num_local_groups; group+=(blockDim.x *blockDim.y))
              tmp_block_phi[group] = block_phi[group + num_groups*nm_offset];

            for (int dd = 0; dd < blockDim.y; ++dd){
//            for (int dd = 0; dd < dd_max; ++dd){

              int dir = dd + d_shift;
              if (dir >= num_directions) break;

              double ell_d_nm =  ell[nm_offset+dir*nidx];
            //  tmp_rhs contains psi
              for (int group = GTID ; group < num_local_groups; group+=(blockDim.x *blockDim.y))
                  tmp_block_phi[group] += tmp_rhs[group + dd*num_local_groups] * ell_d_nm;

            }
            for (int group = GTID ; group < num_local_groups; group+=(blockDim.x *blockDim.y))
               block_phi[group + num_groups*nm_offset] = tmp_block_phi[group];
          }

#else

           for (int dd = 0; dd < blockDim.y; ++dd){
              int dir = dd + (d/blockDim.y)*blockDim.y ;

              if (dir >= num_directions) break;

              for(int nm_offset = 0; nm_offset < nidx; nm_offset += 1){
                double ell_d_nm =  ell[nm_offset+dir*nidx]; 
                for (int group = GTID ; group < num_local_groups; group+=(blockDim.x *blockDim.y))  
                   block_phi[group + num_groups*nm_offset] += tmp_rhs[group + dd*num_local_groups] * ell_d_nm;  
              }
              
           }
#endif
           __syncthreads();

      
       } //endof "for (int d = threadIdx.y; d < num_directions; d += blockDim.y){" 


}
