
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#include "cu_utils.h"


void * get_cudaMallocManaged(size_t size){
   void *data;
   hipMallocManaged(&data, size);
   return data;
}



void * get_cudaMalloc(size_t size){
   void *data;
   hipMalloc(&data, size);
   return data;
}


void * get_cudaMallocHost(size_t size){
   void *data;
   hipHostMalloc(&data, size);
   return data;
}


void  do_cudaMemcpyH2D( void *dst, void * src,  size_t size){
  hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
}

void  do_cudaMemcpyD2H( void *dst, void * src,  size_t size){
  hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
}



