
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#include "cu_utils.h"


int get_cudaGetDeviceCount(){
   int ndev;
   hipGetDeviceCount(&ndev);
   return  ndev;
}

void set_cudaSetDevice(int id){
   hipSetDevice(id);
}

void do_cudaDeviceSynchronize(){
   hipDeviceSynchronize();
} 	


void set_cudaMemZeroAsync( void *ptr,  size_t size){
   hipMemsetAsync(ptr,0,size);
}


void * get_cudaMallocManaged(size_t size){
   void *data;
   hipMallocManaged(&data, size);
   return data;
}



void * get_cudaMalloc(size_t size){
   void *data;
   hipMalloc(&data, size);
   return data;
}


void * get_cudaMallocHost(size_t size){
   void *data;
   hipHostMalloc(&data, size);
   return data;
}


void  do_cudaMemcpyH2D( void *dst, void * src,  size_t size){
  hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
}

void  do_cudaMemcpyD2H( void *dst, void * src,  size_t size){
  hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
}

void  do_cudaMemcpyH2D_Async( void *dst, void * src,  size_t size){
  hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice);
}

void  do_cudaMemcpyD2H_Async( void *dst, void * src,  size_t size){
  hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToHost);
}

#ifdef KRIPKE_USE_CUBLAS
hipblasHandle_t get_cublasHandle(){
     static hipblasHandle_t handle;
     static int handle_FLAG = 0;
     if (handle_FLAG==0){
        hipblasCreate(&handle);
        handle_FLAG=1;
     }      
     return handle;
}
#endif


